#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/kernels/unfusedAttentionKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__inline__ __device__ int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
    return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template <typename T>
__global__ void addQKVBiasIA3Transpose(T* q_out, T* k_out, T* v_out, const T* __restrict q_in,
    const T* __restrict bias_q, const T* __restrict k_in, const T* __restrict bias_k, const T* __restrict v_in,
    const T* __restrict bias_v, const int* ia3_tasks, const T* ia3_key_weights, const T* ia3_value_weights,
    const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id = batch_id * seq_len + word_id;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x)
    {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id = batch_id * (head_num * seq_len * size_per_head) + head_id * seq_len * size_per_head
            + word_id * size_per_head + size_id;
        const int src_id = row_id * n + col_id;

        T q = ldg(&q_in[src_id]);
        q_out[target_id] = add(q, ldg(&bias_q[col_id]));

        T k = add(ldg(&k_in[src_id]), ldg(&bias_k[col_id]));
        if (use_ia3_key)
        {
            k = k * ia3_key_weights[ia3_task * n + col_id];
        }
        k_out[target_id] = k;

        T v = add(ldg(&v_in[src_id]), ldg(&bias_v[col_id]));
        if (use_ia3_value)
        {
            v = v * ia3_value_weights[ia3_task * n + col_id];
        }
        v_out[target_id] = v;
    }
}

template <typename T>
__global__ void QKVIA3Transpose(T* q_out, T* k_out, T* v_out, const T* __restrict q_in, const T* __restrict k_in,
    const T* __restrict v_in, const int* ia3_tasks, const T* __restrict ia3_key_weights,
    const T* __restrict ia3_value_weights, const int batch_size, const int seq_len, const int head_num,
    const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id = batch_id * seq_len + word_id;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x)
    {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id = batch_id * (head_num * seq_len * size_per_head) + head_id * seq_len * size_per_head
            + word_id * size_per_head + size_id;
        const int src_id = row_id * n + col_id;

        q_out[target_id] = ldg(&q_in[src_id]);

        T k = ldg(&k_in[src_id]);
        if (use_ia3_key)
        {
            k = k * ia3_key_weights[ia3_task * n + col_id];
        }
        k_out[target_id] = k;

        T v = ldg(&v_in[src_id]);
        if (use_ia3_value)
        {
            v = v * ia3_value_weights[ia3_task * n + col_id];
        }
        v_out[target_id] = v;
    }
}

template <typename T>
void invokeAddQKVBiasIA3Transpose(T* q_buf, T* k_buf, T* v_buf, T* Q, const T* bias_Q, T* K, const T* bias_K, T* V,
    const T* bias_V, const int batch_size, const int seq_len, const int head_num, const int size_per_head,
    const int* ia3_tasks, const T* ia3_key_weights, const T* ia3_value_weights, hipStream_t stream)
{
    const int k = head_num * size_per_head;
    dim3 grid(batch_size, seq_len);
    bool is_add_bias = bias_Q != nullptr;
    if (sizeof(T) == 4 || k % 2 != 0)
    {
        dim3 block(min(k, 512));
        if (is_add_bias)
        {
            addQKVBiasIA3Transpose<T><<<grid, block, 0, stream>>>(q_buf, k_buf, v_buf, Q, bias_Q, K, bias_K, V, bias_V,
                ia3_tasks, ia3_key_weights, ia3_value_weights, batch_size, seq_len, head_num, size_per_head);
        }
        else
        {
            QKVIA3Transpose<T><<<grid, block, 0, stream>>>(q_buf, k_buf, v_buf, Q, K, V, ia3_tasks, ia3_key_weights,
                ia3_value_weights, batch_size, seq_len, head_num, size_per_head);
        }
        sync_check_cuda_error();
    }
    else
    {
        using T2 = typename TypeConverter<T>::Type; // fp16 to half2, bf16 to bf162
        dim3 block(min(k / 2, 512));
        if (is_add_bias)
        {
            addQKVBiasIA3Transpose<T2><<<grid, block, 0, stream>>>((T2*) q_buf, (T2*) k_buf, (T2*) v_buf, (const T2*) Q,
                (const T2*) bias_Q, (const T2*) K, (const T2*) bias_K, (const T2*) V, (const T2*) bias_V, ia3_tasks,
                (const T2*) ia3_key_weights, (const T2*) ia3_value_weights, batch_size, seq_len, head_num,
                size_per_head / 2);
        }
        else
        {
            QKVIA3Transpose<T2><<<grid, block, 0, stream>>>((T2*) q_buf, (T2*) k_buf, (T2*) v_buf, (const T2*) Q,
                (const T2*) K, (const T2*) V, ia3_tasks, (const T2*) ia3_key_weights, (const T2*) ia3_value_weights,
                batch_size, seq_len, head_num, size_per_head / 2);
        }
        sync_check_cuda_error();
    }
}

#define INSTANTIATE_ADDQKVBIASIA3_TRANSPOSE(T)                                                                         \
    template void invokeAddQKVBiasIA3Transpose(T* q_buf, T* k_buf, T* v_buf, T* Q, const T* bias_Q, T* K,              \
        const T* bias_K, T* V, const T* bias_V, const int batch_size, const int seq_len, const int head_num,           \
        const int size_per_head, const int* ia3_tasks, const T* ia3_key_weights, const T* ia3_value_weights,           \
        hipStream_t stream)
INSTANTIATE_ADDQKVBIASIA3_TRANSPOSE(float);
INSTANTIATE_ADDQKVBIASIA3_TRANSPOSE(half);
#ifdef ENABLE_BF16
INSTANTIATE_ADDQKVBIASIA3_TRANSPOSE(__hip_bfloat16);
#endif
#undef INSTANTIATEADDQKVBIASTRANSPOSE

template <typename T, typename T_IN, int ITEMS_PER_THREAD>
__global__ void softmax_kernel(T* attn_score, const T_IN* qk, const T* attn_mask, const T* linear_bias_slopes,
    const int64_t batch_size, const int64_t head_num, const int64_t q_length, const int64_t k_length,
    const float qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    const int64_t bi = blockIdx.y; // Batch index.
    const int64_t hi = blockIdx.z; // Head index.

    __shared__ float s_mean, s_max;

    const float linear_bias_slope = linear_bias_slopes != nullptr ? (float) linear_bias_slopes[hi] : 0.0f;

    // Loop along with Q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x)
    {
        float data[ITEMS_PER_THREAD];
        float local_max = -1e20f;

        // Loop along with K dimension.
        int64_t ki{threadIdx.x};
        for (int i = 0; ki < k_length; i++, ki += blockDim.x)
        {
            int64_t qk_offset{((bi * head_num + hi) * q_length + qi) * k_length + ki};

            float qk_val = static_cast<float>(qk[qk_offset]);
            float qk_bias = 0.0f;

            if (linear_bias_slopes != nullptr)
            {
                // We don't handle the upper diagonal (ki > qi) separately, whose values
                // are negligible due to the negative infinity mask. And it matches with
                // the HF's implementation.
                qk_bias += static_cast<float>(linear_bias_slope * (ki - qi));
            }

            int64_t mask_offset = ((int64_t) bi * q_length + qi) * k_length + ki;
            float mask_val = static_cast<float>(ldg(&attn_mask[mask_offset]));
            qk_bias += (1.0f - mask_val) * -10000.0f;

            data[i] = qk_scale * qk_val + qk_bias;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0)
        {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length; i++, ki += blockDim.x)
        {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0)
        {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length; i++, ki += blockDim.x)
        {
            int64_t qk_offset{((bi * head_num + hi) * q_length + qi) * k_length + ki};
            attn_score[qk_offset] = (T) (data[i] * s_mean);
        }
    }
}

template <typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2(T* attn_score, const T* qk_buf, const T* attn_mask, const T* linear_bias_slopes,
    const int64_t batch_size, const int64_t head_num, const int64_t q_length, const int64_t k_length, const T qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    T2* attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2 = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2 = reinterpret_cast<const T2*>(attn_mask);

    const int64_t bi = blockIdx.y; // Batch index
    const int64_t hi = blockIdx.z; // Head index.

    __shared__ float s_mean, s_max;

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE = cuda_cast<T2>(1.0f);
    const T2 ZERO = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale_h2 = cuda_cast<T2>(qk_scale);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;
    int64_t k_length_half = k_length / 2;

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x)
    {
        T2 data[ITEMS_PER_THREAD];
        float local_max = -1e20f;

        // Loop over k dimension.
        int64_t ki{threadIdx.x};
        for (int i = 0; ki < k_length_half && i < ITEMS_PER_THREAD; i++, ki += blockDim.x)
        {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int64_t qk_offset{((bi * head_num + hi) * q_length + qi) * k_length_half + ki};
            int64_t mask_offset = (bi * q_length + qi) * k_length_half + ki;

            // The value of QK^T matrix at (qi, ki).
            T2 qk = qk_buf_h2[qk_offset];
            // The bias value to the position (qi, ki) including both mask and positional bias.
            T2 qk_bias = ZERO;

            if (linear_bias_slopes != nullptr)
            {
                // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                // separately, whose values are negligible due to the negative infinity mask.
                T2 dist(2.0f * ki - qi, 2.0f * ki + 1 - qi);
                qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(linear_bias_slope, dist));
            }

            T2 mask_val = ldg(&attn_mask_h2[mask_offset]);
            qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(hsub2<T2>(ONE, mask_val), NEG_INFTY));

            data[i] = hadd2<T2>(hmul2<T2>(qk, qk_scale_h2), qk_bias);
            local_max = fmax(local_max, fmax((float) data[i].x, (float) data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0)
        {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0.0f;
        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length_half && i < ITEMS_PER_THREAD; i++, ki += blockDim.x)
        {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], cuda_cast<T2>(s_max)));
            local_sum += (float) (data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0)
        {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length_half && i < ITEMS_PER_THREAD; i++, ki += blockDim.x)
        {
            int64_t qk_offset{((bi * head_num + hi) * q_length + qi) * k_length_half + ki};
            attn_score_h2[qk_offset] = hmul2<T2>(data[i], cuda_cast<T2>(s_mean));
        }
    }
}

template <typename T, int K_ITEMS_PER_THREAD, int Q_ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2_v2(T* attn_score, const T* qk_buf, const T* attn_mask, const T* linear_bias_slopes,
    const int64_t batch_size, const int64_t head_num, const int64_t q_length, const int64_t k_length, const T scalar)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    // QK^T matrix of shape (batch_size, head_num, q_length, k_length / 2)
    T2* attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2 = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2 = reinterpret_cast<const T2*>(attn_mask);

    const int64_t bi = blockIdx.y; // Batch index
    const int64_t hi = blockIdx.z; // Head index.

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE = cuda_cast<T2>(1.0f);
    const T2 ZERO = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale = cuda_cast<T2>(scalar);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;
    const int64_t k_length_half = k_length / 2;
    __shared__ float s_sum[Q_ITEMS_PER_THREAD], s_max[Q_ITEMS_PER_THREAD];

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x * Q_ITEMS_PER_THREAD)
    {
        T2 data[Q_ITEMS_PER_THREAD][K_ITEMS_PER_THREAD];

        int64_t qk_offset[Q_ITEMS_PER_THREAD];

        float local_max[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++)
        {
            local_max[j] = -1e20f;
        }

        // Loop over k dimension.
        const int64_t q_items = min(static_cast<int64_t>((q_length - qi + gridDim.x - 1) / gridDim.x),
            static_cast<int64_t>(Q_ITEMS_PER_THREAD));
        // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
        int64_t ki{threadIdx.x};
        for (int i = 0; ki < k_length_half && i < K_ITEMS_PER_THREAD; ++i, ki += blockDim.x)
        {

            int64_t mask_offset[Q_ITEMS_PER_THREAD];

            for (int j = 0; j < q_items; j++)
            {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * k_length_half + ki;
                mask_offset[j] = (bi * q_length + qi + j * gridDim.x) * k_length_half + ki;
            }

            T2 mask_val[Q_ITEMS_PER_THREAD];
            for (int j = 0; j < q_items; j++)
            {
                mask_val[j] = ldg(&attn_mask_h2[mask_offset[j]]);
            }

            T2 qk[Q_ITEMS_PER_THREAD];
            for (int j = 0; j < q_items; j++)
            {
                qk[j] = qk_buf_h2[qk_offset[j]];
            }

            T2 pos_bias[Q_ITEMS_PER_THREAD];
            if (linear_bias_slopes != nullptr)
            {
                for (int j = 0; j < q_items; j++)
                {
                    // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                    // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                    // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                    // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                    // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                    // separately, whose values are negligible due to the negative infinity mask.
                    int64_t qidx = qi + j * gridDim.x;
                    T2 dist(2.0f * ki - qidx, 2.0f * ki + 1 - qidx);
                    pos_bias[j] = hmul2<T2>(linear_bias_slope, dist);
                }
            }

            for (int j = 0; j < q_items; j++)
            {
                mask_val[j] = hmul2<T2>(hsub2<T2>(ONE, mask_val[j]), NEG_INFTY);
            }

            for (int j = 0; j < q_items; j++)
            {
                T2 val = hadd2<T2>(hmul2<T2>(qk_scale, qk[j]), mask_val[j]);
                if (linear_bias_slopes != nullptr)
                {
                    val = hadd2<T2>(val, pos_bias[j]);
                }
                data[j][i] = val;
                local_max[j] = fmax(local_max[j], fmax((float) data[j][i].x, (float) data[j][i].y));
            }
        }

        if (blockDim.x <= 32)
        {
            warpReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }
        else
        {
            blockReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }

        if (threadIdx.x == 0)
        {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++)
            {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++)
        {
            local_sum[j] = {0.f};
        }

        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length_half && i < K_ITEMS_PER_THREAD; ++i, ki += blockDim.x)
        {
            for (int j = 0; j < q_items; ++j)
            {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], cuda_cast<T2>(s_max[j])));
            }

            for (int j = 0; j < q_items; j++)
            {
                local_sum[j] += (float) (data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32)
        {
            warpReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }
        else
        {
            blockReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }

        if (threadIdx.x == 0)
        {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++)
            {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        ki = (int64_t) threadIdx.x;
        for (int i = 0; ki < k_length_half && i < K_ITEMS_PER_THREAD; ++i, ki += blockDim.x)
        {
            for (int j = 0; j < q_items; j++)
            {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * k_length_half + ki;
            }

            for (int j = 0; j < q_items; j++)
            {
                attn_score_h2[qk_offset[j]] = hmul2<T2>(data[j][i], cuda_cast<T2>(s_sum[j]));
            }
        }
    }
}

#define LAUNCH_MASKED_SOFTMAX_(T_, ITEMS_PER_THREAD)                                                                   \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    block.x = divUp(block.x, 32) * 32;                                                                                 \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2)                                                                                                      \
    {                                                                                                                  \
        if (grid.x % 4 == 0)                                                                                           \
        {                                                                                                              \
            grid.x /= 4;                                                                                               \
            softmax_kernel_h2_v2<T_, ITEMS_PER_THREAD, 4><<<grid, block, 0, stream>>>((T_*) param.attention_score,     \
                (const T_*) param.qk, (const T_*) param.attention_mask, (const T_*) param.linear_bias_slopes,          \
                param.batch_size, param.num_heads, param.q_length, param.k_length, (const T_) param.qk_scale);         \
        }                                                                                                              \
        else                                                                                                           \
        {                                                                                                              \
            softmax_kernel_h2<T_, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>((T_*) param.attention_score,           \
                (const T_*) param.qk, (const T_*) param.attention_mask, (const T_*) param.linear_bias_slopes,          \
                param.batch_size, param.num_heads, param.q_length, param.k_length, (const T_) param.qk_scale);         \
        }                                                                                                              \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        softmax_kernel<T, T_IN, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>(param.attention_score, param.qk,         \
            param.attention_mask, param.linear_bias_slopes, param.batch_size, param.num_heads, param.q_length,         \
            param.k_length, param.qk_scale);                                                                           \
    }

#define LAUNCH_MASKED_SOFTMAX(ITEMS_PER_THREAD) LAUNCH_MASKED_SOFTMAX_(half, ITEMS_PER_THREAD)

template <typename T, typename T_IN>
void invokeMaskedSoftmax(MaskedSoftmaxParam<T, T_IN>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360)
    {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 32768)
    {
        TLLM_CHECK(false); // Not implemented - it's not clear we want to use the unfused kernel in that case.
    }
    else if (block.x > 16384)
    {
        LAUNCH_MASKED_SOFTMAX(32)
    }
    else if (block.x > 8192)
    {
        LAUNCH_MASKED_SOFTMAX(16)
    }
    else if (block.x > 4096)
    {
        LAUNCH_MASKED_SOFTMAX(8)
    }
    else if (block.x > 2048)
    {
        LAUNCH_MASKED_SOFTMAX(4)
    }
    else if (block.x > 1024)
    {
        LAUNCH_MASKED_SOFTMAX(2)
    }
    else if (block.x > 0)
    {
        LAUNCH_MASKED_SOFTMAX(1)
    }
}

template void invokeMaskedSoftmax(MaskedSoftmaxParam<float, float>& param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, float>& param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, half>& param, hipStream_t stream);

#ifdef ENABLE_BF16
template <>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, float>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T = __hip_bfloat16;
    using T_IN = float;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360)
    {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 32768)
    {
        TLLM_CHECK(false); // Not implemented - it's not clear we want to use the unfused kernel in that case.
    }
    else if (block.x > 16384)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 32)
    }
    else if (block.x > 8192)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 16)
    }
    else if (block.x > 4096)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 8)
    }
    else if (block.x > 2048)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 4)
    }
    else if (block.x > 1024)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 2)
    }
    else if (block.x > 0)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 1)
    }
}

template <>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, __hip_bfloat16>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T = __hip_bfloat16;
    using T_IN = __hip_bfloat16;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360)
    {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 32768)
    {
        TLLM_CHECK(false); // Not implemented - it's not clear we want to use the unfused kernel in that case.
    }
    else if (block.x > 16384)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 32)
    }
    else if (block.x > 8192)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 16)
    }
    else if (block.x > 4096)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 8)
    }
    else if (block.x > 2048)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 4)
    }
    else if (block.x > 1024)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 2)
    }
    else if (block.x > 0)
    {
        LAUNCH_MASKED_SOFTMAX_(__hip_bfloat16, 1)
    }
}

#endif

#undef LAUNCH_MASKED_SOFTMAX
#undef LAUNCH_MASKED_SOFTMAX_

template <typename T>
__global__ void transpose(const T* src, T* dst, const int batch_size, const int seq_len, const int head_num,
    const int size_per_head, const float* scale, int int8_mode)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int batch_id = tid / (head_num * seq_len * size_per_head);
    int head_id = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
    int seq_id = (tid % (seq_len * size_per_head)) / size_per_head;
    int id = tid % size_per_head;

    int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);

    if (int8_mode == 2)
    {
        using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
        using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;

        const Float_Packed_T scale_val = cuda_cast<Float_Packed_T>(*scale);
        reinterpret_cast<Int8_Packed_T*>(dst)[target_id]
            = cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src[tid]) * scale_val);
    }
    else
    {
        dst[target_id] = src[tid];
    }
}

template <>
__global__ void transpose(const float* src, float* dst, const int batch_size, const int seq_len, const int head_num,
    const int size_per_head, const float* scale, int int8_mode)
{
    int batch_id = blockIdx.x / (head_num * seq_len);
    int seq_id = blockIdx.x % seq_len;
    int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;

    const int target_id = batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
        + head_id * size_per_head + threadIdx.x;
    const int src_id = blockIdx.x * size_per_head + threadIdx.x;

    if (int8_mode == 2)
    {
        const float scale_val = *scale;
        reinterpret_cast<int8_t*>(dst)[target_id] = cuda_cast<int8_t>(src[src_id] * scale_val);
    }
    else
    {
        dst[target_id] = src[src_id];
    }
}

template <typename T>
void invokeTransposeQKV(T* dst, T* src, const int batch_size, const int seq_len, const int head_num,
    const int size_per_head, const float* scale, const int int8_mode, hipStream_t stream)
{
    dim3 grid, block;
    if (sizeof(T) == 2)
    {
        int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        while (seq_per_block < 4 && grid.x % 2 == 0)
        {
            grid.x /= 2;
            seq_per_block *= 2;
        }

        TLLM_CHECK(grid.x * seq_per_block == (size_t) batch_size * head_num * seq_len);

        if (seq_per_block * size_per_head % 2 == 0)
        {
            block.x = seq_per_block * size_per_head / 2;
            if (std::is_same<T, half>::value)
            {
                transpose<half2><<<grid, block, 0, stream>>>(
                    (half2*) src, (half2*) dst, batch_size, seq_len, head_num, size_per_head / 2, scale, int8_mode);
            }
#ifdef ENABLE_BF16
            else
            {
                transpose<__hip_bfloat162><<<grid, block, 0, stream>>>((__hip_bfloat162*) src, (__hip_bfloat162*) dst,
                    batch_size, seq_len, head_num, size_per_head / 2, scale, int8_mode);
            }
#endif
        }
        else
        {
            block.x = seq_per_block * size_per_head;
            transpose<T>
                <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
        }
    }
    else
    {
        const int seq_per_block = 1;
        grid.x = batch_size * head_num * seq_len / seq_per_block;
        block.x = seq_per_block * size_per_head;
        transpose<T>
            <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
    }
}

#define INSTANTIATE_TRANSPOSE_QKV(T)                                                                                   \
    template void invokeTransposeQKV(T* src, T* dst, const int batch_size, const int seq_len, const int head_num,      \
        const int size_per_head, const float* scale, const int int8_mode, hipStream_t stream)
INSTANTIATE_TRANSPOSE_QKV(float);
INSTANTIATE_TRANSPOSE_QKV(half);
#ifdef ENABLE_BF16
INSTANTIATE_TRANSPOSE_QKV(__hip_bfloat16);
#endif
#undef INSTANTIATE_TRANSPOSE_QKV

template <typename T>
__global__ void add_QKV_bias_rebuild_padding_ia3(const T* Q, const T* bias_Q, const T* K, const T* bias_K, const T* V,
    const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, const int* ia3_tasks, const T* ia3_key_weights,
    const T* ia3_value_weights, const int batch_size, const int seq_len, const int head_num, const int size_per_head,
    const int* mask_offset)
{
    const int bid = blockIdx.x;

    const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
    const int n = head_num * size_per_head;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[tgt_batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);
    for (int idx = threadIdx.x; idx < n; idx += blockDim.x)
    {
        const int tgt_head_id = idx / size_per_head;
        const int tgt_hidden_id = idx % size_per_head;

        const int src_id = bid * n + idx;
        const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + tgt_head_id * seq_len * size_per_head
            + tgt_seq_id * size_per_head + tgt_hidden_id;

        q_buf_[tgt_id] = add(ldg(&Q[src_id]), ldg(&bias_Q[idx]));

        T k = ldg(&K[src_id]);
        if (use_ia3_key)
        {
            k = k * ia3_key_weights[ia3_task * n + idx];
        }
        k_buf_[tgt_id] = add(k, ldg(&bias_K[idx]));

        T v = ldg(&V[src_id]);
        if (use_ia3_value)
        {
            v = v * ia3_value_weights[ia3_task * n + idx];
        }
        v_buf_[tgt_id] = add(v, ldg(&bias_V[idx]));
    }
}

template <typename T>
__global__ void rebuild_padding_ia3(const T* Q, const T* K, const T* V, T* q_buf_, T* k_buf_, T* v_buf_,
    const int* ia3_tasks, const T* ia3_key_weights, const T* ia3_value_weights, const int batch_size, const int seq_len,
    const int head_num, const int size_per_head, const int* mask_offset)
{
    const int bid = blockIdx.x;

    const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
    const int n = head_num * size_per_head;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[tgt_batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);
    for (int idx = threadIdx.x; idx < n; idx += blockDim.x)
    {
        const int tgt_head_id = idx / size_per_head;
        const int tgt_hidden_id = idx % size_per_head;

        const int src_id = bid * n + idx;
        const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + tgt_head_id * seq_len * size_per_head
            + tgt_seq_id * size_per_head + tgt_hidden_id;

        q_buf_[tgt_id] = ldg(&Q[src_id]);

        T k = ldg(&K[src_id]);
        if (use_ia3_key)
        {
            k = k * ia3_key_weights[ia3_task * n + idx];
        }
        k_buf_[tgt_id] = k;

        T v = ldg(&V[src_id]);
        if (use_ia3_value)
        {
            v = v * ia3_value_weights[ia3_task * n + idx];
        }
        v_buf_[tgt_id] = v;
    }
}

template <typename T>
void invokeAddQKVBiasIA3RebuildPadding(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf,
    T* k_buf, T* v_buf, const int batch_size, const int seq_len, const int head_num, const int size_per_head,
    const int valid_word_num, const int* mask_offset, const int* ia3_tasks, const T* ia3_key_weights,
    const T* ia3_value_weights, hipStream_t stream)
{
#ifdef ENABLE_BF16
    bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
#else
    bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
#endif
    using T2 = typename TypeConverter<T>::Type; // fp16 to half2, bf16 to bf162
    int block_size = head_num * size_per_head;
    if (is_half2)
    {
        while (block_size > 512)
        {
            if (block_size % 2 == 0)
            {
                block_size /= 2;
            }
            else
            {
                is_half2 = false;
                block_size = std::min(block_size, 512);
                break;
            }
        }
    }
    else
    {
        block_size = std::min(block_size, 512);
    }

    if (bias_Q == nullptr && bias_K == nullptr && bias_V == nullptr)
    {
        if (is_half2)
        {
            rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>((T2*) Q, (T2*) K, (T2*) V, (T2*) q_buf,
                (T2*) k_buf, (T2*) v_buf, ia3_tasks, (const T2*) ia3_key_weights, (const T2*) ia3_value_weights,
                batch_size, seq_len, head_num, size_per_head / 2, mask_offset);
        }
        else
        {
            rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>(Q, K, V, q_buf, k_buf, v_buf, ia3_tasks,
                ia3_key_weights, ia3_value_weights, batch_size, seq_len, head_num, size_per_head, mask_offset);
        }
    }
    else if (bias_Q != nullptr && bias_K != nullptr && bias_V != nullptr)
    {
        if (is_half2)
        {
            add_QKV_bias_rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>((T2*) Q, (const T2*) bias_Q,
                (T2*) K, (const T2*) bias_K, (T2*) V, (const T2*) bias_V, (T2*) q_buf, (T2*) k_buf, (T2*) v_buf,
                ia3_tasks, (const T2*) ia3_key_weights, (const T2*) ia3_value_weights, batch_size, seq_len, head_num,
                size_per_head / 2, mask_offset);
        }
        else
        {
            add_QKV_bias_rebuild_padding_ia3<<<valid_word_num, block_size, 0, stream>>>(Q, bias_Q, K, bias_K, V, bias_V,
                q_buf, k_buf, v_buf, ia3_tasks, ia3_key_weights, ia3_value_weights, batch_size, seq_len, head_num,
                size_per_head, mask_offset);
        }
    }
    else
    {
        TLLM_CHECK(false);
    }
}

#define INSTANTIATE_ADDQKVBIASIA3_REBUILD_PADDING(T)                                                                   \
    template void invokeAddQKVBiasIA3RebuildPadding(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V,                \
        const T* bias_V, T* q_buf, T* k_buf, T* v_buf, const int batch_size, const int seq_len, const int head_num,    \
        const int size_per_head, const int valid_word_num, const int* mask_offset, const int* ia3_tasks,               \
        const T* ia3_key_weights, const T* ia3_value_weights, hipStream_t stream)
INSTANTIATE_ADDQKVBIASIA3_REBUILD_PADDING(float);
INSTANTIATE_ADDQKVBIASIA3_REBUILD_PADDING(half);
#ifdef ENABLE_BF16
INSTANTIATE_ADDQKVBIASIA3_REBUILD_PADDING(__hip_bfloat16);
#endif
#undef INSTANTIATEADDQKVBIASREBUILDPADDING

template <typename T>
__global__ void transpose_remove_padding(const T* src, T* dst, const int batch_size, const int seq_len,
    const int head_num, const int size_per_head, const int* mask_offset, const float* scale, const int int8_mode)
{
    // TODO: optimize this kernel?
    // do remove_sequence_length_padding
    const int bid = blockIdx.x; // batch * seq_len or valid_word_num

    const int src_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int src_seq_id = (bid + mask_offset[bid]) % seq_len;

    const int dst_seq_id = bid;

    const int src_offset_base = src_batch_id * seq_len * head_num * size_per_head + src_seq_id * size_per_head;
    const int dst_offset_base = dst_seq_id * head_num * size_per_head;

    using Int8_Packed_T = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    const Float_Packed_T scale_val
        = int8_mode == 2 ? cuda_cast<Float_Packed_T>(*scale) : cuda_cast<Float_Packed_T>(0.0f);

    for (int idx = threadIdx.x; idx < head_num * size_per_head; idx += blockDim.x)
    {
        const int head_id = idx / size_per_head;
        const int hidden_id = idx % size_per_head;
        const T src_elem = ldg(&src[src_offset_base + head_id * seq_len * size_per_head + hidden_id]);
        if (int8_mode == 2)
        {
            reinterpret_cast<Int8_Packed_T*>(dst)[dst_offset_base + idx]
                = cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src_elem) * scale_val);
        }
        else
        {
            dst[dst_offset_base + idx] = src_elem;
        }
    }
}

// clang-format off
 template<typename T>
 void invokeTransposeAttentionOutRemovePadding(T*           src,
                                               T*           dst,
                                               const int    valid_word_num,
                                               const int    batch_size,
                                               const int    seq_len,
                                               const int    head_num,
                                               const int    size_per_head,
                                               const int*   mask_offset,
                                               const float* scale,
                                               const int    int8_mode,
                                               hipStream_t stream)
 {
 #ifdef ENABLE_BF16
     bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
 #else
     bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
 #endif
     using T2       = typename TypeConverter<T>::Type;  // fp16 to half2, bf16 to bf162
     int block_size = head_num * size_per_head;
     if (is_half2) {
         while (block_size > 512) {
             if (block_size % 2 == 0) {
                 block_size /= 2;
             }
             else {
                 is_half2   = false;
                 block_size = std::min(block_size, 1024);
                 break;
             }
         }
     }
     else {
         block_size = std::min(block_size, 1024);
     }

     if (is_half2) {
         transpose_remove_padding<T2><<<valid_word_num, block_size, 0, stream>>>(
             (T2*)src, (T2*)dst, batch_size, seq_len, head_num, size_per_head / 2, mask_offset, scale, int8_mode);
     }
     else {
         transpose_remove_padding<<<valid_word_num, block_size, 0, stream>>>(
             src, dst, batch_size, seq_len, head_num, size_per_head, mask_offset, scale, int8_mode);
     }
 }

// clang-format on

#define INSTANTIATE_TRANSPOSE_ATTENTION_OUT_REMOVE_PADDING(T)                                                          \
    template void invokeTransposeAttentionOutRemovePadding(T* src, T* dst, const int valid_word_num,                   \
        const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int* mask_offset,  \
        const float* scale, const int int8_mode, hipStream_t stream)
INSTANTIATE_TRANSPOSE_ATTENTION_OUT_REMOVE_PADDING(float);
INSTANTIATE_TRANSPOSE_ATTENTION_OUT_REMOVE_PADDING(half);
#ifdef ENABLE_BF16
INSTANTIATE_TRANSPOSE_ATTENTION_OUT_REMOVE_PADDING(__hip_bfloat16);
#endif
#undef INSTANTIATE_TRANSPOSE_ATTENTION_OUT_REMOVE_PADDING

template <typename T, bool ADD_BIAS>
__global__ void add_fusedQKV_bias_transpose_kernel(T* q_buf, T* k_buf, T* v_buf, T* QKV, const T* __restrict qkv_bias,
    const int* seq_lens, const int* padding_offset, const int batch_size, const int seq_len, const int token_num,
    const int head_num, const int kv_head_num, const int size_per_head, const float* scale, const int int8_mode)
{
    //   QKV: [token_num, hidden + 2 * kv_head_num * size_per_head]
    //   qkv_bias: [hidden + 2 * kv_head_num * size_per_head]
    //   q_buf: [batch, head_num, seq_len, size_per_head]
    //   k_buf, v_buf: [batch, kv_head_num, seq_len, size_per_head]
    // For cross attention where q/k/v buffer could be nullptr, writing to split buffer is suppressed when null
    T* qkv_ptr[3] = {q_buf, k_buf, v_buf};
    const bool has_padding = padding_offset == nullptr;
    const int hidden = head_num * size_per_head; // hidden dim Q
    const int n = hidden + 2 * kv_head_num * size_per_head;

    for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < token_num * n; index += gridDim.x * blockDim.x)
    {
        const int bias_id = index % n;

        const int token_idx = index / n;
        const int token_padded_idx = token_idx + (has_padding ? 0 : padding_offset[token_idx]);
        const int target_batch_id = token_padded_idx / seq_len;
        const int actual_seq_len = seq_lens[target_batch_id];
        const int seq_id = token_padded_idx % seq_len;
        const bool valid_seq = seq_id < actual_seq_len || !has_padding;

        int qkv_id;
        int head_id;
        int size_id = index % size_per_head;
        if (kv_head_num < head_num)
        {
            // [token, h + 2*kv_head_num, d]
            //  ^^^^^  ^^^^^^^^
            //    m       n
            // TODO: This block will also work for MHA but
            // would that be slower due to more branches?
            head_id = (index % n) / size_per_head;
            if (head_id < head_num) // Q
            {
                qkv_id = 0;
            }
            else //  K/V
            {
                head_id = head_id - head_num;
                if (head_id < kv_head_num) // K
                {
                    qkv_id = 1;
                }
                else // V
                {
                    qkv_id = 2;
                    head_id = head_id - kv_head_num;
                }
            }
        }
        else
        {
            // [token, 3, h, d]
            //  ^^^^^  ^^^^^^^
            //    m       n
            qkv_id = (index % n) / hidden;
            head_id = (index % hidden) / size_per_head;
        }

        T val = 0.f;
        if (valid_seq)
        {
            if (int8_mode == 2)
            {
                val = cuda_cast<T>(cuda_cast<float>(reinterpret_cast<const int8_t*>(QKV)[index]) * scale[qkv_id]);
            }
            else
            {
                val = ldg(&QKV[index]);
            }
            if (ADD_BIAS)
            {
                val = val + ldg(&qkv_bias[bias_id]);
            }
        }
        // Write to split QKV buffer
        if (head_num == kv_head_num || qkv_id == 0) // QKV or Q when MQA/GQA
        {
            const int target_batch_stride = head_num * seq_len * size_per_head;
            const int target_head_stride = seq_len * size_per_head;
            const int target_seq_stride = size_per_head;
            if (qkv_ptr[qkv_id])
                qkv_ptr[qkv_id][target_batch_id * target_batch_stride + head_id * target_head_stride
                    + seq_id * target_seq_stride + size_id]
                    = val;
        }
        else if (head_num != kv_head_num && qkv_id > 0) // KV when MQA/GQA
        {
            const int target_batch_stride = kv_head_num * seq_len * size_per_head;
            const int target_head_stride = seq_len * size_per_head;
            const int target_seq_stride = size_per_head;
            if (qkv_ptr[qkv_id])
                qkv_ptr[qkv_id][target_batch_id * target_batch_stride + head_id * target_head_stride
                    + seq_id * target_seq_stride + size_id]
                    = val;
        }
    }
}

template <typename T>
struct Vec_t
{
    static constexpr int size = 0;
};

template <>
struct Vec_t<float>
{
    using Type = float2;
    static constexpr int size = 2;
};

template <>
struct Vec_t<half>
{
    using Type = uint32_t;
    static constexpr int size = 2;
};

#ifdef ENABLE_BF16
template <>
struct Vec_t<__hip_bfloat16>
{
    using Type = __hip_bfloat162;
    static constexpr int size = 2;
};
#endif

template <typename T, bool ADD_BIAS>
__global__ void add_fusedQKV_bias_transpose_kernel(T* q_buf, T* k_buf, T* v_buf, T* QKV, const T* __restrict qkv_bias,
    const int* seq_lens, const int* padding_offset, const int batch_size, const int seq_len, const int head_num,
    const int kv_head_num, const int size_per_head, const int rotary_embedding_dim, float rotary_embedding_base,
    RotaryScalingType const rotary_scale_type, float rotary_embedding_scale, const int rotary_embedding_max_positions,
    PositionEmbeddingType const position_embedding_type)
{
    // This kernel add bias to QKV, which has shape [batch_size, seq_len, 3, head_num, size_per_head], and
    // QKV split to 3 split buffer q, k, v and transpose them to [batch_size, head_num, seq_len, size_per_head].
    // For q and k, also apply the rotary embedding.
    // For cross attention where q/k/v buffer could be nullptr, writing to split buffer is suppressed when null

    // NOTE:
    // head_num == kv_head_num
    //   QKV src shape (batch_size, seq_len, 3, head_num, size_per_head)
    //                  ^^^^^^^^^^^^^^^^^^^  ^^^^^^^^^^^^^^^^^^^^^^^^^^
    //                           m                        n
    //   QKV dst shape (3, batch_size, head_num, seq_len, size_per_head)
    // head_num != kv_head_num
    //   QKV src shape: (batch_size, seq_len, head_num * size_per_head + 2 * kv_head_num * size_per_head)
    //                   ^^^^^^^^^^^^^^^^^^^  ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
    //                             m                               n
    //   Q dst shape: (batch_size, head_num, seq_len, size_per_head)
    //   KV dst shape: (batch_size, kv_head_num, seq_len, size_per_head)
    extern __shared__ __align__(sizeof(float2)) char smem_[]; // align on largest vector type

    constexpr int vec_size = Vec_t<T>::size;
    using Vec_t = typename Vec_t<T>::Type;
    const int token_idx = blockIdx.x;
    const int token_padding_offset = (padding_offset == nullptr || token_idx < 0) ? 0 : padding_offset[token_idx];
    const int tgt_token_idx = token_idx + token_padding_offset;
    const bool has_padding = padding_offset == nullptr;

    const int batch_idx = tgt_token_idx / seq_len;
    const int seq_idx = tgt_token_idx % seq_len;
    const int actual_seq_len = seq_lens[batch_idx];
    const bool valid_seq = seq_idx < actual_seq_len || !has_padding;

    const int head_idx = blockIdx.y;
    const int tidx = threadIdx.x;

    const int total_seq_len = seq_len;

    const bool is_seq_masked = !valid_seq;
    const bool is_head_size_masked = tidx * vec_size >= size_per_head;
    const bool is_masked = is_head_size_masked || is_seq_masked;

    const int hidden_size = head_num * size_per_head;
    const int hidden_idx = head_idx * size_per_head + tidx * vec_size;
    const int qheads_per_kv_head = head_num / kv_head_num;
    const int kv_head_idx = head_idx / qheads_per_kv_head;
    const int hidden_idx_kv = kv_head_idx * size_per_head + tidx * vec_size;
    const int n = (head_num + 2 * kv_head_num) * size_per_head;

    const int dst_kv_seq_idx = seq_idx;
    const int src_k_offset = hidden_size;
    const int src_v_offset = hidden_size + kv_head_num * size_per_head;

    // NOTE: q has seq len excluding prefix prompt
    // head_num == kv_head_num:
    //   src QKV: [batch, time, 3, head_num, size_per_head]
    // head_num != kv_head_num:
    //   src QKV: [batch, time, head_num * size_per_head + 2 * kv_head_num * size_per_head]
    const int src_q_idx = token_idx * n + hidden_idx;
    const int src_k_idx = token_idx * n + src_k_offset + hidden_idx_kv;
    const int src_v_idx = token_idx * n + src_v_offset + hidden_idx_kv;

    // destination offset.
    const int dest_q_idx = batch_idx * size_per_head * seq_len * head_num + head_idx * size_per_head * seq_len
        + seq_idx * size_per_head + tidx * vec_size;

    const int dest_kv_idx = batch_idx * size_per_head * total_seq_len * kv_head_num
        + kv_head_idx * size_per_head * total_seq_len + dst_kv_seq_idx * size_per_head + tidx * vec_size;

    Vec_t q, k, v, zero;
    Vec_t q_bias, k_bias, v_bias;
    if (valid_seq)
    {
        mmha::update_rotary_base_n_scale(rotary_embedding_base, rotary_embedding_scale, rotary_scale_type,
            rotary_embedding_dim, rotary_embedding_max_positions, actual_seq_len);
    }

#pragma unroll
    for (int i = 0; i < sizeof(Vec_t) / sizeof(uint32_t); i++)
    {
        reinterpret_cast<uint32_t*>(&zero)[i] = 0u;
    }

    // load q,k,v and add bias
    if (!is_masked)
    {
        q = *reinterpret_cast<const Vec_t*>(&QKV[src_q_idx]);
        k = *reinterpret_cast<const Vec_t*>(&QKV[src_k_idx]);
        v = *reinterpret_cast<const Vec_t*>(&QKV[src_v_idx]);

        if (ADD_BIAS)
        {
            q_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx]);
            k_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx_kv + src_k_offset]);
            v_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx_kv + src_v_offset]);

            q = mmha::add(q, q_bias);
            k = mmha::add(k, k_bias);
            v = mmha::add(v, v_bias);
        }
    }

    switch (position_embedding_type)
    {
    case PositionEmbeddingType::kROPE_GPTJ:
    {
        mmha::apply_rotary_embedding(
            q, k, tidx, rotary_embedding_dim, rotary_embedding_base, rotary_embedding_scale, dst_kv_seq_idx);
        break;
    }
    case PositionEmbeddingType::kROPE_GPT_NEOX:
    {
        const bool do_rotary = !is_masked && vec_size * tidx < rotary_embedding_dim;

        T* q_smem = reinterpret_cast<T*>(smem_);
        T* k_smem = q_smem + rotary_embedding_dim;

        const int half_rotary_dim = rotary_embedding_dim / 2;
        const int half_idx = (tidx * vec_size) / half_rotary_dim;
        const int intra_half_idx = (tidx * vec_size) % half_rotary_dim;
        const int smem_pitch = half_rotary_dim; // TODO: adjust for bank conflicts?

        if (do_rotary)
        {
            *reinterpret_cast<Vec_t*>(q_smem + half_idx * smem_pitch + intra_half_idx) = q;
            *reinterpret_cast<Vec_t*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
        }

        __syncthreads();

        const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
        constexpr int tidx_factor = vec_size / 2;
        if (do_rotary)
        {
            mmha::vec_from_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
            mmha::vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);

            mmha::apply_rotary_embedding(q, k, transpose_idx / tidx_factor, rotary_embedding_dim, rotary_embedding_base,
                rotary_embedding_scale, dst_kv_seq_idx);

            mmha::write_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
            mmha::write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        }

        __syncthreads();

        if (do_rotary)
        {
            q = *reinterpret_cast<Vec_t*>(q_smem + half_idx * smem_pitch + intra_half_idx);
            k = *reinterpret_cast<Vec_t*>(k_smem + half_idx * smem_pitch + intra_half_idx);
        }
        break;
    }
    }
    if (!is_masked)
    {

        if (q_buf)
            *reinterpret_cast<Vec_t*>(&q_buf[dest_q_idx]) = q;

        if ((head_num == kv_head_num) || (head_idx == (kv_head_idx * qheads_per_kv_head)))
        {
            // we always need the following writes for KV cache
            if (k_buf)
                *reinterpret_cast<Vec_t*>(&k_buf[dest_kv_idx]) = k;
            if (v_buf)
                *reinterpret_cast<Vec_t*>(&v_buf[dest_kv_idx]) = v;
        }
    }
    else if (is_seq_masked && !is_head_size_masked)
    {
        // Set padding to zero in case of potential nan generated.
        if (q_buf)
            *reinterpret_cast<Vec_t*>(&q_buf[dest_q_idx]) = zero;

        if ((head_num == kv_head_num) || (head_idx == (kv_head_idx * qheads_per_kv_head)))
        {
            // we always need the following writes for KV cache
            if (k_buf)
                *reinterpret_cast<Vec_t*>(&k_buf[dest_kv_idx]) = zero;
            if (v_buf)
                *reinterpret_cast<Vec_t*>(&v_buf[dest_kv_idx]) = zero;
        }
    }
}

#define FUSED_QKV_BIAS_TRANSPOSE_LAUNCH(T, ADD_BIAS)                                                                   \
    add_fusedQKV_bias_transpose_kernel<T, ADD_BIAS><<<grid, block, 0, stream>>>(q_buf, k_buf, v_buf, QKV, qkv_bias,    \
        seq_lens, padding_offset, batch_size, seq_len, token_num, head_num, kv_head_num, size_per_head, scale,         \
        int8_mode);

#define FUSED_QKV_BIAS_ROTARY_TRANSPOSE_LAUNCH(T, ADD_BIAS)                                                            \
    add_fusedQKV_bias_transpose_kernel<T, ADD_BIAS><<<grid, block, smem_size, stream>>>(q_buf, k_buf, v_buf, QKV,      \
        qkv_bias, seq_lens, padding_offset, batch_size, seq_len, head_num, kv_head_num, size_per_head,                 \
        rotary_embedding_dim, rotary_embedding_base, rotary_scale_type, rotary_embedding_scale,                        \
        rotary_embedding_max_positions, position_embedding_type);

template <typename T>
void invokeAddFusedQKVBiasTranspose(T* q_buf, T* k_buf, T* v_buf, T* QKV, const T* qkv_bias, const int* seq_lens,
    const int* padding_offset, const int batch_size, const int seq_len, const int token_num, const int head_num,
    const int kv_head_num, const int size_per_head, const int rotary_embedding_dim, const float rotary_embedding_base,
    const RotaryScalingType rotary_scale_type, const float rotary_embedding_scale,
    const int rotary_embedding_max_positions, const PositionEmbeddingType position_embedding_type, const float* scale,
    const int int8_mode, hipStream_t stream)
{
    // [bs, seq_len, 3, head, Dh]
    if (rotary_embedding_dim == 0)
    {
        const int m = token_num;
        const int n = head_num * size_per_head;
        dim3 block(384);
        dim3 grid((int) (ceil(1.0 * m * n / 384)));

        if (qkv_bias != nullptr)
        {
            FUSED_QKV_BIAS_TRANSPOSE_LAUNCH(T, true);
        }
        else
        {
            FUSED_QKV_BIAS_TRANSPOSE_LAUNCH(T, false);
        }
    }
    else
    {
        TLLM_CHECK_WITH_INFO(int8_mode != 2, "w8a8 not yet implemented with RoPE"); // TODO
        // To implement rotary embeddings, each thread processes two QKV elems:
        dim3 block((size_per_head / Vec_t<T>::size + 31) / 32 * 32);
        dim3 grid(token_num, head_num);
        size_t smem_size
            = (position_embedding_type == PositionEmbeddingType::kROPE_GPT_NEOX ? 2 * rotary_embedding_dim * sizeof(T)
                                                                                : 0);
        // NOTE: add offset for rotary embedding
        if (qkv_bias != nullptr)
        {
            FUSED_QKV_BIAS_ROTARY_TRANSPOSE_LAUNCH(T, true);
        }
        else
        {
            FUSED_QKV_BIAS_ROTARY_TRANSPOSE_LAUNCH(T, false);
        }
    }
}

#define INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(T)                                                                       \
    template void invokeAddFusedQKVBiasTranspose(T* q_buf, T* k_buf, T* v_buf, T* QKV, const T* qkv_bias,              \
        const int* seq_lens, const int* padding_offset, const int batch_size, const int seq_len, const int token_num,  \
        const int head_num, const int kv_head_num, const int size_per_head, const int rotary_embedding_dim,            \
        const float rotary_embedding_base, const RotaryScalingType rotary_scale_type,                                  \
        const float rotary_embedding_scale, const int rotary_embedding_max_poisitions,                                 \
        const PositionEmbeddingType position_embedding_type, const float* scale, const int int8_mode,                  \
        hipStream_t stream)
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(float);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half);
#ifdef ENABLE_BF16
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(__hip_bfloat16);
#endif
#undef INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE

template <typename T>
__global__ void transpose_4d(T* dst, T* src, const int dim0, const int dim1, const int dim2, const int dim3,
    const int dim0_leading_dim, const int ite)
{
    // transpose from [dim0, dim1, dim2, dim3] to [dim2, X, dim1, dim3]
    // where the dimension of X is dim0_leading_dim, and offset is ite * dim0
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < dim0 * dim1 * dim2 * dim3; i += blockDim.x * gridDim.x)
    {
        int index = i;
        const int d3 = index % dim3;
        index = (index - d3) / dim3;
        const int d2 = index % dim2;
        index = (index - d2) / dim2;
        const int d1 = index % dim1;
        index = (index - d1) / dim1;
        const int d0 = index % dim0;
        index = (index - d0) / dim0;
        dst[d2 * dim0_leading_dim * dim1 * dim3 + (d0 + dim0 * ite) * dim1 * dim3 + d1 * dim3 + d3] = src[i];
    }
}

template <>
__global__ void transpose_4d(half* dst, half* src, const int dim0, const int dim1, const int dim2, const int dim3,
    const int dim0_leading_dim, const int ite)
{
    half2* dst_ptr = (half2*) dst;
    half2* src_ptr = (half2*) src;
    const int half_dim3 = dim3 / 2;
    // transpose from [dim0, dim1, dim2, half_dim3] to [dim2, dim0, dim1, half_dim3]
    // where the dimension of X is dim0_leading_dim, and offset is ite * dim0
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < dim0 * dim1 * dim2 * half_dim3; i += blockDim.x * gridDim.x)
    {
        int index = i;
        const int d3 = index % half_dim3;
        index = (index - d3) / half_dim3;
        const int d2 = index % dim2;
        index = (index - d2) / dim2;
        const int d1 = index % dim1;
        index = (index - d1) / dim1;
        const int d0 = index % dim0;
        index = (index - d0) / dim0;
        dst_ptr[d2 * dim0_leading_dim * dim1 * half_dim3 + (d0 + dim0 * ite) * dim1 * half_dim3 + d1 * half_dim3 + d3]
            = src_ptr[i];
    }
}

template <typename T>
void invokeTranspose4d(T* dst, T* src, const int local_batch_size, const int seq_len, const int size_per_head,
    const int local_hidden_units, const int local_head_num, const int batch_size, const int ite, hipStream_t stream)
{
    transpose_4d<<<local_batch_size * seq_len * local_hidden_units / 512, 512 / (4 / (sizeof(T))), 0, stream>>>(
        dst, src, local_batch_size, local_head_num, seq_len, size_per_head, batch_size, ite);
}

#define INSTANTIATE_TRANSPOSE_4D(T)                                                                                    \
    template void invokeTranspose4d(T* dst, T* src, const int local_batch_size, const int seq_len,                     \
        const int size_per_head, const int local_hidden_units, const int local_head_num, const int batch_size,         \
        const int ite, hipStream_t stream)
INSTANTIATE_TRANSPOSE_4D(float);
INSTANTIATE_TRANSPOSE_4D(half);
#undef INSTANTIATE_TRANSPOSE_4D

template <typename T, typename T_cache, typename KVCacheBuffer>
__global__ void transpose4dBatchMajorKVCache(const T* kSrc, const T* vSrc, KVCacheBuffer kvCacheBuffer,
    const int headNum, const int sizePerHead, const int seqLen, const int attentionWindowSize,
    const float* kvScaleOrigQuant, const int* sequence_lengths)
{
    // We allow only fp32/fp16/bf16 as input types
    static_assert(sizeof(T) == 4 || sizeof(T) == 2, "");

    constexpr int X_ELEMS = (sizeof(T) == 4) ? 4 : 8;
    constexpr bool ENABLE_8BITS_CACHE = sizeof(T_cache) == 1;
    using T_dst = T_cache;
    using T_src = typename mmha::packed_type<T, X_ELEMS>::type;

    const int batchIdx = blockIdx.y;
    const int headIdx = blockIdx.z;

    // idx is over output dimension L * sizePerHead / x for values
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // threadIdx.y 0 handles k, while threadIdx.y 1 handles v.
    const bool handle_k = (threadIdx.y == 0);
    const int sizePerHeadDivX = sizePerHead / X_ELEMS;

    if (idx >= sizePerHeadDivX * seqLen)
    {
        return;
    }

    // Get linear token index
    int tokenIdx = idx / sizePerHeadDivX;
    // Apply cyclic kv cache if tokenIdx >= max_attention_window_size.
    // which means we will drop the tokens in the beginning if seqLen > max_attention_window_size.
    const int tokenIdxLowerBound = max(sequence_lengths[batchIdx] - attentionWindowSize, 0);
    // Get channel index
    const int channelIdx = idx % sizePerHeadDivX;
    if (tokenIdx >= sequence_lengths[batchIdx] || tokenIdx < tokenIdxLowerBound)
    {
        return;
    }

    // Get token index in kv cache
    auto tokenKVIdx = kvCacheBuffer.getKVTokenIdx(tokenIdx);
    // Get pointer to the dst block given sequence, head and token ids
    auto valDst = handle_k ? reinterpret_cast<T_dst*>(kvCacheBuffer.getKBlockPtr(batchIdx, tokenKVIdx))
                           : reinterpret_cast<T_dst*>(kvCacheBuffer.getVBlockPtr(batchIdx, tokenKVIdx));

    // Local to block dst idx
    int inBlockIdx = kvCacheBuffer.getKVLocalIdx(tokenKVIdx, headIdx, sizePerHeadDivX, channelIdx);

    // 16 byte loads will handle "x" dimension
    const size_t srcOffset = (batchIdx * headNum + headIdx) * sizePerHead * seqLen;
    auto valSrc = reinterpret_cast<const T_src*>((handle_k ? kSrc : vSrc) + srcOffset);

    T_src val = valSrc[idx];
    if (ENABLE_8BITS_CACHE)
    {
        // If T is fp32, T_src is float4 and mmha::num_elems<T_src>::value returns 4
        // If T is fp16/bf16, T_src is uint4 and mmha::num_elems<T_src>::value returns 8
        // mmha::packed_type<int8_t ...>::type becomes uint32_t or uint64_t respectively
        // FIXME mmha::num_elems semantic is confusing
        inBlockIdx = inBlockIdx * sizeof(mmha::packed_type<T_dst, mmha::num_elems<T_src>::value>::type);
        // Cast float scale to dst data type.
        using T_scale = typename mmha::kv_cache_scale_type_t<T, T_cache>::Type;
        T_scale scaleOrigQuant;
        mmha::convert_from_float(&scaleOrigQuant, kvScaleOrigQuant[0]);
        // Store 8bits kv cache.
        mmha::store_8bits_kv_cache_vec(valDst, val, inBlockIdx, scaleOrigQuant);
    }
    else
    {
        reinterpret_cast<T_src*>(valDst)[inBlockIdx] = val;
    }
}

template <typename T, typename KVCacheBuffer>
void invokeTranspose4dBatchMajor(const T* kSrc, const T* vSrc, KVCacheBuffer& kvTable, const int localBatchSize,
    const int seqLen, const int attentionWindowSize, const int sizePerHead, const int localHeadNum,
    const KvCacheDataType cache_type, const float* kvScaleOrigQuant, const int* sequence_lengths, hipStream_t stream)
{
    // Block handles both K and V tile.
    dim3 blockSz(128, 2);
    constexpr int x = (sizeof(T) == 4) ? 4 : 8;
    dim3 gridSz((seqLen * sizePerHead / x + blockSz.x - 1) / blockSz.x, localBatchSize, localHeadNum);

    TLLM_CHECK_WITH_INFO(sizePerHead % x == 0, "Size per head is not a multiple of X");

    if (cache_type == KvCacheDataType::INT8)
    {
        transpose4dBatchMajorKVCache<T, int8_t, KVCacheBuffer><<<gridSz, blockSz, 0, stream>>>(kSrc, vSrc, kvTable,
            localHeadNum, sizePerHead, seqLen, attentionWindowSize, kvScaleOrigQuant, sequence_lengths);
    }
#ifdef ENABLE_FP8
    else if (cache_type == KvCacheDataType::FP8)
    {
        transpose4dBatchMajorKVCache<T, __hip_fp8_e4m3_fnuz, KVCacheBuffer><<<gridSz, blockSz, 0, stream>>>(kSrc, vSrc,
            kvTable, localHeadNum, sizePerHead, seqLen, attentionWindowSize, kvScaleOrigQuant, sequence_lengths);
    }
#endif // ENABLE_FP8
    else
    {
        transpose4dBatchMajorKVCache<T, T, KVCacheBuffer><<<gridSz, blockSz, 0, stream>>>(kSrc, vSrc, kvTable,
            localHeadNum, sizePerHead, seqLen, attentionWindowSize, kvScaleOrigQuant, sequence_lengths);
    }
}

#define INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR_KV_CACHE_TYPE(T, KVCacheBuffer)                                           \
    template void invokeTranspose4dBatchMajor(const T* kSrc, const T* vSrc, KVCacheBuffer& kvTable,                    \
        const int localBatchSize, const int seqLen, const int attentionWindowSize, const int sizePerHead,              \
        const int localHeadNum, const KvCacheDataType cache_type, const float* kvScaleOrigQuant,                       \
        const int* sequence_lengths, hipStream_t stream)

#define INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR(T)                                                                        \
    INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR_KV_CACHE_TYPE(T, KVBlockArray);                                               \
    INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR_KV_CACHE_TYPE(T, KVLinearBuffer);

INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR(float)
INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR(half)
#ifdef ENABLE_BF16
INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR(__hip_bfloat16);
#endif

#undef INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR_KV_CACHE_TYPE
#undef INSTANTIATE_TRANSPOSE_4D_BATCH_MAJOR

template <typename T, typename BT>
__global__ void addRelativeAttentionBiasUnaligned(T* qk_buf, const BT* relative_attention_bias, const int batch_size,
    const int head_num, const int seq_len, int max_seq_len, bool implicit, int num_buckets, int max_distance,
    bool bidirectional)
{
    const int seq_i = blockIdx.x;
    const int batch_id = blockIdx.y / head_num;
    const int head_id = blockIdx.y % head_num;
    const int rel_attn_table_stride = num_buckets; // num_buckets could be modified below, save it beforehand

    for (int seq_j = threadIdx.x; seq_j < seq_len; seq_j += blockDim.x)
    {

        const int qk_index
            = batch_id * head_num * seq_len * seq_len + head_id * seq_len * seq_len + seq_i * seq_len + seq_j;

        if (implicit)
        {
            // compute bias value on the fly (see bert_preprocess_kernels.cu::buildRelativeAttentionBias)
            int relative_buckets = 0;
            int relative_position = seq_j - seq_i;
            if (bidirectional)
            { // special logic in T5 relative attention, both encoder & decoder use this, because rel pos bias is
                // pre-computed once and passed around
                num_buckets /= 2;
                relative_buckets += relative_position > 0 ? num_buckets : 0;
            }
            relative_position = abs(relative_position);

            int max_exact = num_buckets / 2;
            bool is_small = relative_position < max_exact;
            int relative_position_if_large = max_exact
                + (int) (logf(relative_position * 1.0f / max_exact) / logf((float) max_distance / max_exact)
                    * (num_buckets - max_exact));
            relative_position_if_large = min(relative_position_if_large, num_buckets - 1);
            relative_buckets += is_small ? relative_position : relative_position_if_large;
            BT rel_attn_bias = relative_attention_bias[head_id * rel_attn_table_stride + relative_buckets];
            qk_buf[qk_index] = (T) add((T) rel_attn_bias, qk_buf[qk_index]);
        }
        else
        {
            const int bias_index = head_id * max_seq_len * max_seq_len + seq_i * max_seq_len + seq_j;
            qk_buf[qk_index] = (T) add((T) relative_attention_bias[bias_index], qk_buf[qk_index]);
        }
    }
}

template <typename T, typename BT>
void invokeAddRelativeAttentionBiasUnaligned(T* qk_buf, const BT* relative_attention_bias, const int batch_size,
    const int head_num, const int seq_len, const int max_seq_len, hipStream_t stream, bool implicit, int num_buckets,
    int max_distance, bool bidirectional)
{
    // qk_buf: [batch_size, head_num, seq_len, seq_len]
    // relative_attention_bias: [1, head_num, max_seq_len, max_seq_len]
    dim3 grid(seq_len, batch_size * head_num); // increase block parallelism for long sequence scenario
    dim3 block(1024);

    addRelativeAttentionBiasUnaligned<<<grid, block, 0, stream>>>(qk_buf, relative_attention_bias, batch_size, head_num,
        seq_len, max_seq_len, implicit, num_buckets, max_distance, bidirectional);
}

#define INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(T, BT)                                                       \
    template void invokeAddRelativeAttentionBiasUnaligned(T* qk_buf, const BT* relative_attention_bias,                \
        const int batch_size, const int head_num, const int seq_len, const int max_seq_len, hipStream_t stream,       \
        bool implicit, int num_buckets, int max_distance, bool bidirectional)
INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(float, float);
INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(half, half);
INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(float, half);
#ifdef ENABLE_BF16
INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(__hip_bfloat16, __hip_bfloat16);
INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED(float, __hip_bfloat16);
#endif
#undef INSTANTIATE_ADD_RELATIVE_ATTENTION_BIAS_UNALIGNED

template <typename T, typename T_cache, typename KVCacheBuffer>
__global__ void shiftKCache(KVCacheBuffer kvCacheBuffer, KVLinearBuffer shiftKCacheBuffer, const int sizePerHead,
    const int timestep, const int beam_width, const int maxKCacheLen, const int sinkTokenLen,
    const float* kScaleQuantOrig, const int* sequence_lengths, const int* input_lengths, const int rotary_embedding_dim,
    float rotary_embedding_base, RotaryScalingType const rotary_scale_type, float rotary_embedding_scale,
    const int rotary_embedding_max_positions, PositionEmbeddingType const position_embedding_type)
{
    // We allow only fp32/fp16/bf16 as the data types to apply rotary
    static_assert(sizeof(T) == 4 || sizeof(T) == 2, "");
    // Use 8bit cache.
    static constexpr bool ENABLE_8BITS_CACHE = sizeof(T_cache) == 1;
    // FP8 KV Cache.
    static constexpr bool FP8_K_CACHE = std::is_same<T_cache, __hip_fp8_e4m3_fnuz>::value;
    // INT8 KV Cache.
    static constexpr bool INT8_K_CACHE = std::is_same<T_cache, int8_t>::value;

    extern __shared__ __align__(sizeof(float2)) char smem_[]; // align on largest vector type
    // Each thread will handle 16 bytes.
    constexpr int vec_size = 16u / sizeof(T);
    using Vec_k = typename mmha::packed_type<T, vec_size>::type;
    using Vec_k_cache = typename mmha::packed_type<T_cache, vec_size>::type;
    using T_dst = T;
    const int sizePerHeadDivX = sizePerHead / vec_size;

    // The start token idx for the cyclic part in k cache
    const int cyclic_k_cache_start_idx
        = (timestep <= maxKCacheLen) ? sinkTokenLen : sinkTokenLen + timestep - maxKCacheLen;
    // The token idx
    int token_idx
        = (kvCacheBuffer.isSinkToken(blockIdx.x)) ? blockIdx.x : cyclic_k_cache_start_idx + blockIdx.x - sinkTokenLen;
    // The position idx
    const int token_pos_idx = blockIdx.x;
    // Head
    const int head_idx = blockIdx.y;
    // The batch beam idx
    const int batch_beam_idx = blockIdx.z;
    // The beam idx
    const int beam_idx = batch_beam_idx % beam_width;
    // Thread idx
    const int tidx = threadIdx.x;

    // The actual sequence length excluding the paddings.
    // minus 1 because it includes the current timestep while tlength denotes the past token length.
    const int tlength = sequence_lengths[batch_beam_idx] - 1;
    // The context length
    const int inlength = input_lengths[batch_beam_idx];
    // The k cache valid token length
    const int cache_length = (tlength > maxKCacheLen) ? maxKCacheLen : tlength;
    // Mask out the tokens exceed the real total length and tokens in the context phase with beam_idx>0
    const bool valid_seq = token_idx < tlength && !(token_idx < inlength && beam_idx > 0);
    const bool is_head_size_masked = tidx * vec_size >= sizePerHead;

    // Dequant scales for 8bits k cache
    float k_scale_quant_orig = (ENABLE_8BITS_CACHE ? kScaleQuantOrig[0] : 1.0f);

    if (!valid_seq || is_head_size_masked)
    {
        return;
    }

    mmha::update_rotary_base_n_scale(rotary_embedding_base, rotary_embedding_scale, rotary_scale_type,
        rotary_embedding_dim, rotary_embedding_max_positions, cache_length);

    // Get token index in kv cache
    auto token_kv_idx = kvCacheBuffer.getKVTokenIdx(token_idx);

    // Read k cache
    Vec_k k;
    Vec_k_cache k_cache;
    T_cache* k_cache_batch = reinterpret_cast<T_cache*>(kvCacheBuffer.getKBlockPtr(batch_beam_idx, token_kv_idx));
    int inBlockIdx_r = kvCacheBuffer.getKVLocalIdx(token_kv_idx, head_idx, sizePerHead, tidx * vec_size);
    k_cache = *reinterpret_cast<const Vec_k_cache*>(&k_cache_batch[inBlockIdx_r]);
    if constexpr (INT8_K_CACHE)
    {
        using Packed_Float_t = typename mmha::packed_type<float, vec_size>::type;
        mmha::convert_from_float(
            &k, mmha::mul<Packed_Float_t, float>(k_scale_quant_orig, mmha::float_from_int8(k_cache)));
    }
#ifdef ENABLE_FP8
    else if constexpr (FP8_K_CACHE)
    {
        mmha::convert_from_8bit_kv_cache<Vec_k_cache, Vec_k, T_cache, float>(&k, k_cache, k_scale_quant_orig);
    }
#endif // ENABLE_FP8
    else
    {
        k = k_cache;
    }

    // Apply position embedding
    switch (position_embedding_type)
    {
    case PositionEmbeddingType::kROPE_GPTJ:
    {
        mmha::apply_rotary_embedding(
            k, tidx, rotary_embedding_dim, rotary_embedding_base, rotary_embedding_scale, token_pos_idx);
        break;
    }
    case PositionEmbeddingType::kROPE_GPT_NEOX:
    {
        const bool do_rotary = vec_size * tidx < rotary_embedding_dim;

        T* k_smem = reinterpret_cast<T*>(smem_);

        const int half_rotary_dim = rotary_embedding_dim / 2;
        const int half_idx = (tidx * vec_size) / half_rotary_dim;
        const int intra_half_idx = (tidx * vec_size) % half_rotary_dim;
        const int smem_pitch = half_rotary_dim; // TODO: adjust for bank conflicts?

        if (do_rotary)
        {
            *reinterpret_cast<Vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
        }

        __syncthreads();

        const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
        constexpr int tidx_factor = vec_size / 2;
        if (do_rotary)
        {
            mmha::vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
            mmha::apply_rotary_embedding(k, transpose_idx / tidx_factor, rotary_embedding_dim, rotary_embedding_base,
                rotary_embedding_scale, token_pos_idx);
            mmha::write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        }

        __syncthreads();

        if (do_rotary)
        {
            k = *reinterpret_cast<Vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx);
        }
        break;
    }
    }

    // Write k cache
    auto token_k_idx = shiftKCacheBuffer.getKVTokenIdx(token_idx);
    T_dst* kDst = reinterpret_cast<T_dst*>(shiftKCacheBuffer.getKBlockPtr(batch_beam_idx, token_k_idx));
    int inBlockIdx_w = shiftKCacheBuffer.getKVLocalIdx(token_k_idx, head_idx, sizePerHeadDivX, tidx);
    reinterpret_cast<Vec_k*>(kDst)[inBlockIdx_w] = k;
}

template <typename T, typename KVCacheBuffer>
void invokeShiftKCache(KVCacheBuffer kvCacheBuffer, KVLinearBuffer shiftKCacheBuffer, const KvCacheDataType cache_type,
    const int sizePerHead, const int timestep, const int batch_beam, const int kv_head_num, const int beam_width,
    const int maxKCacheLen, const int sinkTokenLen, const float* kScaleQuantOrig, const int* sequence_lengths,
    const int* input_lengths, const int rotary_embedding_dim, float rotary_embedding_base,
    RotaryScalingType const rotary_scale_type, float rotary_embedding_scale, const int rotary_embedding_max_positions,
    PositionEmbeddingType const position_embedding_type, hipStream_t stream)
{
    // Block handles K tile.
    const int token_num_in_k = (timestep <= maxKCacheLen) ? timestep : maxKCacheLen;
    const int vec_size = 16u / sizeof(T);
    dim3 block((sizePerHead / vec_size + 31) / 32 * 32);
    dim3 grid(token_num_in_k, kv_head_num, batch_beam);
    size_t smem_size
        = (position_embedding_type == PositionEmbeddingType::kROPE_GPT_NEOX ? 2 * rotary_embedding_dim * sizeof(T) : 0);

    if (cache_type == KvCacheDataType::INT8)
    {
        shiftKCache<T, int8_t, KVCacheBuffer><<<grid, block, smem_size, stream>>>(kvCacheBuffer, shiftKCacheBuffer,
            sizePerHead, timestep, beam_width, maxKCacheLen, sinkTokenLen, kScaleQuantOrig, sequence_lengths,
            input_lengths, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type, rotary_embedding_scale,
            rotary_embedding_max_positions, position_embedding_type);
    }
#ifdef ENABLE_FP8
    else if (cache_type == KvCacheDataType::FP8)
    {
        shiftKCache<T, __hip_fp8_e4m3_fnuz, KVCacheBuffer><<<grid, block, smem_size, stream>>>(kvCacheBuffer,
            shiftKCacheBuffer, sizePerHead, timestep, beam_width, maxKCacheLen, sinkTokenLen, kScaleQuantOrig,
            sequence_lengths, input_lengths, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type,
            rotary_embedding_scale, rotary_embedding_max_positions, position_embedding_type);
    }
#endif // ENABLE_FP8
    else
    {
        shiftKCache<T, T, KVCacheBuffer><<<grid, block, smem_size, stream>>>(kvCacheBuffer, shiftKCacheBuffer,
            sizePerHead, timestep, beam_width, maxKCacheLen, sinkTokenLen, kScaleQuantOrig, sequence_lengths,
            input_lengths, rotary_embedding_dim, rotary_embedding_base, rotary_scale_type, rotary_embedding_scale,
            rotary_embedding_max_positions, position_embedding_type);
    }
}

#define INSTANTIATE_SHIFT_K_CACHE_CACHE_TYPE(T, KVCacheBuffer)                                                         \
    template void invokeShiftKCache<T, KVCacheBuffer>(KVCacheBuffer kvCacheBuffer, KVLinearBuffer shiftKCacheBuffer,   \
        const KvCacheDataType cache_type, const int sizePerHead, const int timestep, const int batch_beam,             \
        const int kv_head_num, const int beam_width, const int maxKCacheLen, const int sinkTokenLen,                   \
        const float* kScaleQuantOrig, const int* sequence_lengths, const int* input_lengths,                           \
        const int rotary_embedding_dim, float rotary_embedding_base, RotaryScalingType const rotary_scale_type,        \
        float rotary_embedding_scale, const int rotary_embedding_max_positions,                                        \
        PositionEmbeddingType const position_embedding_type, hipStream_t stream)

#define INSTANTIATE_SHIFT_K_CACHE(T)                                                                                   \
    INSTANTIATE_SHIFT_K_CACHE_CACHE_TYPE(T, KVBlockArray);                                                             \
    INSTANTIATE_SHIFT_K_CACHE_CACHE_TYPE(T, KVLinearBuffer);

INSTANTIATE_SHIFT_K_CACHE(float)
INSTANTIATE_SHIFT_K_CACHE(uint16_t)
#ifdef ENABLE_BF16
INSTANTIATE_SHIFT_K_CACHE(__hip_bfloat16);
#endif

#undef INSTANTIATE_SHIFT_K_CACHE_CACHE_TYPE
#undef INSTANTIATE_SHIFT_K_CACHE

} // namespace kernels
} // namespace tensorrt_llm
