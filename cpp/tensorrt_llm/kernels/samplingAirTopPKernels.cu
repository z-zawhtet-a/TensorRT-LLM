#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include <cuda/std/limits>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
template <typename T, typename IdxT, typename AccT>
struct alignas(128) Counter
{
    // Address for input value and index
    T const* in;
    IdxT const* inIdx;

    // The original length of the input
    IdxT oriLen;

    // We are processing the values in multiple passes, from most significant to least
    // significant. In each pass, we keep the length of input (`len`) and the `sum` of
    // current pass, and update them at the end of the pass.
    AccT sum;
    IdxT len;
    float p;

    //  `previousLen` is the length of input in previous pass. Note that `previousLen`
    //  rather than `len` is used for the filtering step because filtering is indeed for
    //  previous pass.
    IdxT previousLen;

    // We determine the bits of the k_th value inside the mask processed by the pass. The
    // already known bits are stored in `kthValueBits`. It's used to discriminate a
    // element is a result (written to `out`), a candidate for next pass (written to
    // `outBuf`), or not useful (discarded). The bits that are not yet processed do not
    // matter for this purpose.
    typename hipcub::Traits<T>::UnsignedBits kthValueBits;

    // Record how many elements have passed filtering. It's used to determine the position
    // in the `outBuf` where an element should be written.
    alignas(128) IdxT filterCnt;

    // For a row inside a batch, we may launch multiple thread blocks. This counter is
    // used to determine if the current block is the last running block. If so, this block
    // will execute scan() and chooseBucket().
    alignas(128) unsigned int finishedBlockCnt;
};

/*******************************Functions*********************************/
using WideT = float4;

#ifdef __CUDA_ARCH__
using ::atomicAdd;

inline __device__ size_t atomicAdd(size_t* address, size_t value)
{
    static_assert(sizeof(size_t) == sizeof(unsigned long long int));
    return atomicAdd((unsigned long long int*) address, (unsigned long long int) value);
}
#endif

//! \brief Provide a ceiling division operation ie. ceil(a / b)
//! \tparam IntType supposed to be only integers for now!
template <typename IntType>
constexpr __host__ __device__ IntType ceilDiv(IntType a, IntType b)
{
    return (a + b - 1) / b;
}

//! \brief Provide an alignment function ie. ceil(a / b) * b
//! \tparam IntType supposed to be only integers for now!
template <typename IntType>
constexpr __host__ __device__ IntType alignTo(IntType a, IntType b)
{
    return ceilDiv(a, b) * b;
}

//! \brief Calcute the number of buckets based on the number of bits per pass.
//! \tparam BitsPerPass. If BitsPerPass==11, the number of buckets is 2048. If BitsPerPass==8, the number of buckets is
//! 256.
template <int BitsPerPass>
__host__ __device__ int constexpr calcNumBuckets()
{
    return 1 << BitsPerPass;
}

//! \brief Calcute the number of passes based on the number of bits per pass.
//! \tparam BitsPerPass. If BitsPerPass==11, the number of passes is 3. If BitsPerPass==8, the number of passes is 4.
template <typename T, int BitsPerPass>
__host__ __device__ int constexpr calcNumPasses()
{
    return ceilDiv<int>(sizeof(T) * 8, BitsPerPass);
}

/**
 * This implementation processes input from the most to the least significant bit (Bit 0 is the least
 * significant (rightmost)). This way, we can skip some passes in the end at the cost of having an unsorted output.
 */
template <typename T, int BitsPerPass>
__device__ int constexpr calcsStartBit(int pass)
{
    int startBit = static_cast<int>(sizeof(T) * 8) - (pass + 1) * BitsPerPass;
    if (startBit < 0)
    {
        startBit = 0;
    }
    return startBit;
}

template <typename T, int BitsPerPass>
__device__ unsigned constexpr calcMask(int pass)
{
    static_assert(BitsPerPass <= 31);
    int numBits = calcsStartBit<T, BitsPerPass>(pass - 1) - calcsStartBit<T, BitsPerPass>(pass);
    return (1 << numBits) - 1;
}

/**
 * Use CUB to twiddle bits.
 */
template <typename T>
__device__ typename hipcub::Traits<T>::UnsignedBits twiddleIn(T key, bool selectMin)
{
    auto bits = reinterpret_cast<typename hipcub::Traits<T>::UnsignedBits&>(key);
    bits = hipcub::Traits<T>::TwiddleIn(bits);
    if (!selectMin)
    {
        bits = ~bits;
    }
    return bits;
}

template <typename T>
__device__ T twiddleOut(typename hipcub::Traits<T>::UnsignedBits bits, bool selectMin)
{
    if (!selectMin)
    {
        bits = ~bits;
    }
    bits = hipcub::Traits<T>::TwiddleOut(bits);
    return reinterpret_cast<T&>(bits);
}

/**
 * Find the bucket based on the radix
 */
template <typename T, int BitsPerPass>
__device__ int calcBucket(T x, int startBit, unsigned mask, bool selectMin)
{
    static_assert(BitsPerPass <= sizeof(int) * 8 - 1, "BitsPerPass is too large that the result type could not be int");
    return (twiddleIn(x, selectMin) >> startBit) & mask;
}

/**
 * This function calculate the bufLen, which is the size of buffer.
 * When the number of candidates for next pass exceeds the bufLen, we choose not to store the candidates. Otherwise, we
 * will load candidates from the original input data.
 */
template <typename T, typename IdxT>
__host__ __device__ IdxT calcBufLen(IdxT len)
{
    // This ratio is calculated based on the element number.
    // If we choose to write the buffers, it means (sizeof(T)+sizeof(IdxT))*bufLen bytes of storing and loading.
    // To ensure we do not access more than len*sizeof(T) bytes. bufLen should be smaller than:
    // len*sizeof(T)/2*(sizeof(T) + sizeof(IdxT)) = len/(2 + sizeof(IdxT) * 2 / sizeof(T))).
    IdxT constexpr ratio = 2 + sizeof(IdxT) * 2 / sizeof(T);
    // Even such estimation is too conservative (due to the global coalescing access). So based on our experiments, we
    // further decrease bufLen by 1/8
    IdxT bufLen = len / (ratio * 8);

    // Align the address to 256 bytes
    bufLen = alignTo(bufLen, 256);
    return bufLen;
}

/**
 * Use ping-pong buffer and set the inBuf and outBuf based on the pass value.
 */
template <typename T, typename IdxT>
__host__ __device__ void setBufPointers(T const* in, IdxT const* inIdx, T* buf1, IdxT* idxBuf1, T* buf2, IdxT* idxBuf2,
    int pass, T const*& inBuf, IdxT const*& inIdxBuf, T*& outBuf, IdxT*& outIdxBuf)
{
    if (pass == 0)
    {
        inBuf = in;
        inIdxBuf = nullptr;
        outBuf = nullptr;
        outIdxBuf = nullptr;
    }
    else if (pass == 1)
    {
        inBuf = in;
        inIdxBuf = inIdx;
        outBuf = buf1;
        outIdxBuf = idxBuf1;
    }
    else if (pass % 2 == 0)
    {
        inBuf = buf1;
        inIdxBuf = idxBuf1;
        outBuf = buf2;
        outIdxBuf = idxBuf2;
    }
    else
    {
        inBuf = buf2;
        inIdxBuf = idxBuf2;
        outBuf = buf1;
        outIdxBuf = idxBuf1;
    }
}

//! \brief Map a Func over the input data, using vectorized load instructions if possible.
//! \tparam T element type
//! \tparam IdxT indexing type
//! \tparam Func void (T x, IdxT idx)
//! \param threadRank rank of the calling thread among all participating threads
//! \param numThreads number of the threads that participate in processing
//! \param in the input data
//! \param len the number of elements to read
//! \param f the lambda taking two arguments (T x, IdxT idx)
template <typename T, typename IdxT, typename Func>
__device__ void vectorizedProcess(size_t threadRank, size_t numThreads, T const* in, IdxT len, Func f)
{
    int constexpr WARP_SIZE = 32;
    if constexpr (sizeof(T) >= sizeof(WideT))
    {
        for (IdxT i = threadRank; i < len; i += numThreads)
        {
            f(in[i], i);
        }
    }
    else
    {
        static_assert(sizeof(WideT) % sizeof(T) == 0);
        int constexpr itemsPerScalar = sizeof(WideT) / sizeof(T);

        // TODO: it's UB
        union
        {
            WideT scalar;
            T array[itemsPerScalar];
        } wide;

        int skipCnt = (reinterpret_cast<size_t>(in) % sizeof(WideT))
            ? ((sizeof(WideT) - reinterpret_cast<size_t>(in) % sizeof(WideT)) / sizeof(T))
            : 0;
        if (skipCnt > len)
        {
            skipCnt = len;
        }
        WideT const* inCast = reinterpret_cast<decltype(inCast)>(in + skipCnt);
        IdxT const lenCast = (len - skipCnt) / itemsPerScalar;

        for (IdxT i = threadRank; i < lenCast; i += numThreads)
        {
            wide.scalar = inCast[i];
            IdxT const real_i = skipCnt + i * itemsPerScalar;
#pragma unroll
            for (int j = 0; j < itemsPerScalar; ++j)
            {
                f(wide.array[j], real_i + j);
            }
        }

        static_assert(WARP_SIZE >= itemsPerScalar);
        // and because itemsPerScalar > skipCnt, WARP_SIZE > skipCnt
        // no need to use loop
        if (threadRank < skipCnt)
        {
            f(in[threadRank], threadRank);
        }
        // because lenCast = (len - skipCnt) / itemsPerScalar,
        // lenCast * itemsPerScalar + itemsPerScalar > len - skipCnt;
        // and so
        // len - (skipCnt + lenCast * itemsPerScalar) < itemsPerScalar <=
        // WARP_SIZE no need to use loop
        IdxT const remain_i = skipCnt + lenCast * itemsPerScalar + threadRank;
        if (remain_i < len)
        {
            f(in[remain_i], remain_i);
        }
    }
}

/**
 * Fused filtering of the current pass and building histogram for the next pass (see steps 4 & 1 in `airTopPSsampling`
 * description).
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass>
__device__ __forceinline__ void filterAndHistogram(T const* inBuf, IdxT const* inIdxBuf, T* outBuf, IdxT* outIdxBuf,
    int previousLen, Counter<T, IdxT, AccT>* counter, AccT* histogram, IdxT* countHistogram, int pass,
    float* outputLogProbs, float* cumLogProbs, IdxT** ids, IdxT const* endIds, IdxT* sequenceLengths,
    FinishedState* finishedOutput, int const batchId, bool earlyStop)
{
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    bool constexpr selectMin = false;
    __shared__ AccT histogramSmem[numBuckets];
    __shared__ IdxT countHistogramSmem[numBuckets];
    for (IdxT i = threadIdx.x; i < numBuckets; i += blockDim.x)
    {
        histogramSmem[i] = 0;
        countHistogramSmem[i] = 0;
    }
    __syncthreads();

    int const startBit = calcsStartBit<T, BitsPerPass>(pass);
    unsigned const mask = calcMask<T, BitsPerPass>(pass);

    if (pass == 0)
    {
        // Passed to vectorizedProcess, this function executes in all blocks in
        // parallel, i.e. the work is split along the input (both, in batches and
        // chunks of a single row). Later, the histograms are merged using
        // atomicAdd.
        auto f = [selectMin, startBit, mask](T value, IdxT)
        {
            int bucket = calcBucket<T, BitsPerPass>(value, startBit, mask, selectMin);

            atomicAdd(histogramSmem + bucket, static_cast<T>(value));
            atomicAdd(countHistogramSmem + bucket, static_cast<IdxT>(1));
        };
        vectorizedProcess(static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x,
            static_cast<size_t>(blockDim.x) * gridDim.x, inBuf, previousLen, f);
    }
    else
    {
        IdxT* pFilterCnt = &counter->filterCnt;
        auto const kthValueBits = counter->kthValueBits;
        int const previousStartBit = calcsStartBit<T, BitsPerPass>(pass - 1);

        // See the remark above on the distributed execution of `f` using
        // vectorizedProcess.
        auto f = [inIdxBuf, outBuf, outIdxBuf, selectMin, startBit, mask, previousStartBit, kthValueBits, pFilterCnt,
                     outputLogProbs, cumLogProbs, ids, endIds, sequenceLengths, finishedOutput, batchId,
                     earlyStop](T value, IdxT i)
        {
            auto const previousBits = (twiddleIn(value, selectMin) >> previousStartBit) << previousStartBit;
            if (previousBits == kthValueBits)
            {
                if (earlyStop)
                {
                    int const currentStep = sequenceLengths[batchId];
                    IdxT index = inIdxBuf ? inIdxBuf[i] : i;
                    ids[batchId][currentStep] = index;
                    epilogue(
                        value, index, outputLogProbs, cumLogProbs, endIds, sequenceLengths, finishedOutput, batchId);
                }
                if (outBuf)
                {
                    IdxT pos = atomicAdd(pFilterCnt, static_cast<IdxT>(1));
                    outBuf[pos] = value;
                    outIdxBuf[pos] = inIdxBuf ? inIdxBuf[i] : i;
                }

                int bucket = calcBucket<T, BitsPerPass>(value, startBit, mask, selectMin);
                atomicAdd(histogramSmem + bucket, static_cast<T>(value));
                atomicAdd(countHistogramSmem + bucket, static_cast<IdxT>(1));
            }
        };
        vectorizedProcess(static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x,
            static_cast<size_t>(blockDim.x) * gridDim.x, inBuf, previousLen, f);
    }

    __syncthreads();
    if (earlyStop)
    {
        return;
    }

    // merge histograms produced by individual blocks
    for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
    {
        if (histogramSmem[i] != 0)
        {
            atomicAdd(histogram + i, histogramSmem[i]);
        }
        if (countHistogramSmem[i] != 0)
        {
            atomicAdd(countHistogram + i, countHistogramSmem[i]);
        }
    }
}

/**
 *  Replace histogram with its own prefix sum (step 2 in `airTopPSsampling` description)
 */
template <typename IdxT, int BitsPerPass, int BlockSize>
__device__ void scan(volatile IdxT* histogram)
{
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    if constexpr (numBuckets >= BlockSize)
    {
        static_assert(numBuckets % BlockSize == 0);
        int constexpr itemsPerThread = numBuckets / BlockSize;
        typedef hipcub::BlockLoad<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoad;
        typedef hipcub::BlockStore<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_STORE_TRANSPOSE> BlockStore;
        typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;

        __shared__ union
        {
            typename BlockLoad::TempStorage load;
            typename BlockScan::TempStorage scan;
            typename BlockStore::TempStorage store;
        } tempStorage;

        IdxT threadData[itemsPerThread];

        BlockLoad(tempStorage.load).Load(histogram, threadData);
        __syncthreads();

        BlockScan(tempStorage.scan).InclusiveSum(threadData, threadData);
        __syncthreads();

        BlockStore(tempStorage.store).Store(histogram, threadData);
    }
    else
    {
        typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;
        __shared__ typename BlockScan::TempStorage tempStorage;

        IdxT threadData = 0;
        if (threadIdx.x < numBuckets)
        {
            threadData = histogram[threadIdx.x];
        }

        BlockScan(tempStorage).InclusiveSum(threadData, threadData);
        __syncthreads();

        if (threadIdx.x < numBuckets)
        {
            histogram[threadIdx.x] = threadData;
        }
    }
}

/**
 * Calculate in which bucket the k-th value will fall
 *  (steps 3 in `airTopPSsampling` description)
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass>
__device__ void chooseBucket(
    Counter<T, IdxT, AccT>* counter, AccT const* histogram, IdxT const* countHistogram, AccT const sum, int const pass)
{
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
    {
        AccT prev = (i == 0) ? 0 : histogram[i - 1];
        AccT cur = histogram[i];

        // one and only one thread will satisfy this condition, so counter is
        // written by only one thread
        if ((prev < sum && cur >= sum) || (sum <= 0 && i == 0))
        {
            counter->sum = sum - prev;        // how many values still are there to find
            counter->len = countHistogram[i]; // cur - prev; // number of values in next pass
            typename hipcub::Traits<T>::UnsignedBits bucket = i;
            int startBit = calcsStartBit<T, BitsPerPass>(pass);
            counter->kthValueBits |= bucket << startBit;
        }
    }
}

/**
 * Computes sequenceLength, finished state, outputLogProbs, and cumLogProbs.
 */
template <typename T, typename IdxT>
__device__ void epilogue(T const value, IdxT const index, float* outputLogProbs, float* cumLogProbs, IdxT const* endIds,
    IdxT* sequenceLengths, FinishedState* finishedOutput, int const batchId)
{
    if (outputLogProbs != nullptr || cumLogProbs != nullptr)
    {
        float res = logf(value);
        if (outputLogProbs)
        {
            outputLogProbs[batchId] = res;
        }
        if (cumLogProbs)
        {
            cumLogProbs[batchId] += res;
        }
    }
    if (index == endIds[batchId])
    {
        if (finishedOutput != nullptr)
        {
            finishedOutput[batchId].setFinishedEOS();
        }
        // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be outputted
    }
    else
    {
        // We don't need to set output finished state as it is assumed to be in non finished state
        sequenceLengths[batchId] += 1;
    }
}

/**
 *  Find the target element.
 *  (steps 4 in `airTopPSsampling` description)
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass>
__device__ void lastFilter(T const* inBuf, IdxT const* inIdxBuf, IdxT currentLen, Counter<T, IdxT, AccT>* counter,
    float* outputLogProbs, float* cumLogProbs, IdxT** ids, IdxT const* endIds, IdxT* sequenceLengths,
    FinishedState* finishedOutput, int const batchId)
{
    auto const kthValueBits = counter->kthValueBits;
    auto const equalValue = twiddleOut<T>(kthValueBits, false);
    int const currentStep = sequenceLengths[batchId];
    IdxT* outIdx = &ids[batchId][currentStep];
    if (threadIdx.x == 0)
    {
        *outIdx = cuda::std::numeric_limits<IdxT>::max();
    }
    __syncthreads();

    for (IdxT i = threadIdx.x; i < currentLen; i += blockDim.x)
    {
        if (inBuf[i] == equalValue)
        {
            atomicMin(outIdx, inIdxBuf ? inIdxBuf[i] : i);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        epilogue(equalValue, *outIdx, outputLogProbs, cumLogProbs, endIds, sequenceLengths, finishedOutput, batchId);
    }
}

/******************************Kernel**********************************/
/**
 * We call this parallel top-p algorithm AIR Top-P, because this method is based on our previous work called AIR Top-K.
 * Details about AIR Top-K can be found here https://dl.acm.org/doi/10.1145/3581784.360706, the open-source code is here
 * https://github.com/rapidsai/raft/blob/main/cpp/include/raft/matrix/detail/select_radix.cuh
 *
 * It is expected to call this kernel multiple times (passes), in each pass we process a radix,
 * going from the most significant towards the least significant bits (MSD).
 *
 * Conceptually, each pass consists of 4 steps:
 *
 * 1. Calculate histogram
 *      First, transform bits into a digit, the value of which is in the range
 *      [0, 2^{BITS_PER_PASS}-1]. Then count the frequency of each digit value along with the summation of corresponding
 * elements and the result is a countHistogram and histogram. That is, countHistogram[i] contains the count of inputs
 * having value i.
 *
 * 2. Scan the histogram
 *      Inclusive prefix sum is computed for the histogram. After this step, histogram[i] contains
 *      the prefix-sum of inputs having value <= i.
 *
 * 3. Find the bucket j of the histogram that just exceed the p*total_sum value falls into
 *
 * 4. Filtering
 *      Input elements whose digit value <j are the top-p elements. Since the k-th value must be in
 *      the bucket j, we write all elements in bucket j into a intermediate buffer out_buf. For the
 *      next pass, these elements are used as input, and we update the counter->sum accordingly. T
 *
 * In the implementation, the filtering step is delayed to the next pass so the filtering and
 * histogram computation are fused. In this way, inputs are read once rather than twice.
 *
 * During the filtering step, we won't write candidates (elements in bucket j) to `out_buf` if the
 * number of candidates is larger than the length of `out_buf` (this could happen when the leading
 * bits of input values are almost the same). And then in the next pass, inputs are read from `in`
 * rather than from `in_buf`. The benefit is that we can save the cost of writing candidates and
 * their indices.
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass, int BlockSize, bool is_fused_filter = false>
__global__ void airTopPSsampling(Counter<T, IdxT, AccT>* counters, AccT* histograms, IdxT* countHistograms, IdxT** ids,
    int* sequenceLengths, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, IdxT const* endIds, int const batchSize, bool const* skipDecode, int const pass, T* buf1,
    IdxT* idxBuf1, T* buf2, IdxT* idxBuf2)
{
    assert(sequenceLengths != nullptr);
    int const tid = threadIdx.x;
    int const batchId = blockIdx.y;
    auto counter = counters + batchId;

    // Skip kernel if this sampling method is not chosen
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchId] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchId]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    // Exit early if sequence has finished
    if (finishState.isFinished())
    {
        if (pass == 0 && tid == 0)
        {
            if (finishedOutput != nullptr)
            {
                finishedOutput[batchId] = finishState;
            }
            ids[batchId][sequenceLengths[batchId]] = endIds[batchId];
        }
        return;
    }

    /// Set length
    AccT currentSum;
    IdxT previousLen;
    IdxT currentLen;

    if (pass == 0)
    {
        currentSum = 0;
        previousLen = counter->len;
        // Need to do this so setting counter->previousLen for the next pass is correct.
        // This value is meaningless for pass 0, but it's fine because pass 0 won't be the
        // last pass in this implementation so pass 0 won't hit the "if (pass ==
        // numPasses - 1)" branch.
        currentLen = counter->len;
    }
    else
    {
        currentSum = counter->sum;
        currentLen = counter->len;
        previousLen = counter->previousLen;
    }
    if (currentLen == 0)
    {
        return;
    }
    bool const earlyStop = (currentLen == 1);
    IdxT const bufLen = calcBufLen<T>(counter->oriLen);

    /// Set address
    T const* inBuf = nullptr;
    IdxT const* inIdxBuf = nullptr;
    T* outBuf = nullptr;
    IdxT* outIdxBuf = nullptr;

    setBufPointers(counter->in, counter->inIdx, buf1 + bufLen * batchId, idxBuf1 + bufLen * batchId,
        buf2 + bufLen * batchId, idxBuf2 + bufLen * batchId, pass, inBuf, inIdxBuf, outBuf, outIdxBuf);

    // "previousLen > bufLen" means previous pass skips writing buffer
    if (pass == 0 || pass == 1 || previousLen > bufLen)
    {
        inBuf = counter->in;
        inIdxBuf = counter->inIdx;
        previousLen = counter->oriLen;
    }
    // "currentLen > bufLen" means current pass will skip writing buffer
    if (pass == 0 || currentLen > bufLen)
    {
        outBuf = nullptr;
        outIdxBuf = nullptr;
    }
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    auto histogram = histograms + batchId * numBuckets;
    auto countHistogram = countHistograms + batchId * numBuckets;

    filterAndHistogram<T, IdxT, AccT, BitsPerPass>(inBuf, inIdxBuf, outBuf, outIdxBuf, previousLen, counter, histogram,
        countHistogram, pass, outputLogProbs, cumLogProbs, ids, endIds, sequenceLengths, finishedOutput, batchId,
        earlyStop);

    __syncthreads();

    bool isLastBlock = false;
    if (threadIdx.x == 0)
    {
        unsigned int finished = atomicInc(&counter->finishedBlockCnt, gridDim.x - 1);
        isLastBlock = (finished == (gridDim.x - 1));
    }

    if (__syncthreads_or(isLastBlock))
    {
        if (earlyStop)
        {
            return;
        }
        scan<AccT, BitsPerPass, BlockSize>(histogram);
        __syncthreads();
        if (pass == 0)
        {
            currentSum = histogram[numBuckets - 1] * counter->p;
        }
        __syncthreads();

        chooseBucket<T, IdxT, AccT, BitsPerPass>(counter, histogram, countHistogram, currentSum, pass);
        __syncthreads();

        int constexpr numPasses = calcNumPasses<T, BitsPerPass>();
        // reset for next pass
        if (pass != numPasses - 1)
        {
            for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
            {
                histogram[i] = 0;
                countHistogram[i] = 0;
            }
        }
        if (threadIdx.x == 0)
        {
            counter->previousLen = currentLen;
            // not necessary for the last pass, but put it here anyway
            counter->filterCnt = 0;
        }

        if (pass == numPasses - 1)
        {
            if constexpr (is_fused_filter)
            {
                lastFilter<T, IdxT, AccT, BitsPerPass>(outBuf ? outBuf : inBuf, outIdxBuf ? outIdxBuf : inIdxBuf,
                    outBuf ? currentLen : counter->oriLen, counter, outputLogProbs, cumLogProbs, ids, endIds,
                    sequenceLengths, finishedOutput, batchId);

                __syncthreads();
            }
        }
    }
}

/**
 * Initialize the Counter<T, IdxT, AccT> and the histogram and countHistogram.
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass, int BlockSize>
__global__ void airTopPInitialize(Counter<T, IdxT, AccT>* counters, int const batchSize, int const len, T const* in,
    IdxT const* inIdx, float const topP, float const* topPs, hiprandState_t* curandstate, AccT* histograms,
    IdxT* countHistograms)
{
    auto const batchIdx = blockIdx.x;
    Counter<T, IdxT, AccT>* counter = counters + batchIdx;
    IdxT offset = batchIdx * len;
    IdxT bufOffset = batchIdx * calcBufLen<T>(len);
    if (threadIdx.x == 0)
    {
        counter->in = in + offset;
        counter->inIdx = nullptr;
        if (inIdx)
        {
            counter->inIdx = inIdx + offset;
        }

        counter->len = len;
        counter->oriLen = len;
        counter->previousLen = len;

        float const probThreshold = (topPs != nullptr) ? topPs[batchIdx] : topP;
        float const randP = hiprand_uniform(curandstate + batchIdx) * probThreshold;
        counter->p = randP;
        counter->sum = 0;

        counter->kthValueBits = 0;
        counter->finishedBlockCnt = 0;
        counter->filterCnt = 0;
    }

    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    AccT* histogram = histograms + batchIdx * numBuckets;
    for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
    {
        histogram[i] = 0;
    }

    IdxT* countHistogram = nullptr;
    if (countHistograms)
    {
        countHistogram = countHistograms + batchIdx * numBuckets;
        for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
        {
            countHistogram[i] = 0;
        }
    }
}

/*
 *  Calculate the number of blocks based on the batchSize and len to avoid tailing effect.
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass, int BlockSize>
unsigned calcAirTopPBlockNum(int batchSize, IdxT len, int smCnt)
{
    int constexpr VECTORIZED_READ_SIZE = 16;
    static_assert(VECTORIZED_READ_SIZE / sizeof(T) >= 1);

    int activeBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &activeBlocks, airTopPSsampling<T, IdxT, AccT, BitsPerPass, BlockSize, false>, BlockSize, 0);
    activeBlocks *= smCnt;

    IdxT bestNumBlocks = 0;
    float bestTailWavePenalty = 1.0f;
    IdxT const maxNumBlocks = ceilDiv<IdxT>(len, VECTORIZED_READ_SIZE / sizeof(T) * BlockSize);
    for (int numWaves = 1;; ++numWaves)
    {
        IdxT numBlocks = std::min(maxNumBlocks, static_cast<IdxT>(std::max(numWaves * activeBlocks / batchSize, 1)));
        IdxT itemsPerThread = ceilDiv<IdxT>(len, numBlocks * BlockSize);
        itemsPerThread = alignTo<IdxT>(itemsPerThread, VECTORIZED_READ_SIZE / sizeof(T));
        numBlocks = ceilDiv<IdxT>(len, itemsPerThread * BlockSize);
        float actualNumWaves = static_cast<float>(numBlocks) * batchSize / activeBlocks;
        float tailWavePenalty = (ceilf(actualNumWaves) - actualNumWaves) / ceilf(actualNumWaves);

        // 0.15 is determined experimentally. It also ensures breaking the loop
        // early, e.g. when numWaves > 7, tailWavePenalty will always <0.15
        if (tailWavePenalty < 0.15)
        {
            bestNumBlocks = numBlocks;
            break;
        }
        else if (tailWavePenalty < bestTailWavePenalty)
        {
            bestNumBlocks = numBlocks;
            bestTailWavePenalty = tailWavePenalty;
        }

        if (numBlocks == maxNumBlocks)
        {
            break;
        }
    }
    return bestNumBlocks;
}

template <typename T>
void invokeBatchAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    T const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds,
    float const maxTopP, float const* topPs, hipStream_t stream, int blockNum, bool const* skipDecode)
{
    using IdxT = int;
    using AccT = float;
    IdxT const vocabSize = vocabSizePadded;
    int constexpr BitsPerPass = 11;
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    IdxT const bufLen = calcBufLen<T>(vocabSize);

    int constexpr SAMPLING_BLOCK_SIZE = 512;
    int constexpr THREADS_PER_CTA_TOP_P_INIT = 1024;

    Counter<T, IdxT, AccT>* counters = nullptr;
    AccT* histograms = nullptr;
    IdxT* countHistograms = nullptr;
    T* buf1 = nullptr;
    IdxT* idxBuf1 = nullptr;
    T* buf2 = nullptr;
    IdxT* idxBuf2 = nullptr;
    std::vector<size_t> sizes = {sizeof(*counters) * batchSize, sizeof(*histograms) * numBuckets * batchSize,
        sizeof(*countHistograms) * numBuckets * batchSize, sizeof(*buf1) * bufLen * batchSize,
        sizeof(*idxBuf1) * bufLen * batchSize, sizeof(*buf2) * bufLen * batchSize,
        sizeof(*idxBuf2) * bufLen * batchSize};
    size_t totalSize = calcAlignedSize(sizes);
    if (workspace == nullptr)
    {
        workspaceSize = totalSize;
        return;
    }
    std::vector<void*> alignedPointers;
    calcAlignedPointers(alignedPointers, workspace, sizes);
    counters = static_cast<decltype(counters)>(alignedPointers[0]);
    histograms = static_cast<decltype(histograms)>(alignedPointers[1]);
    countHistograms = static_cast<decltype(countHistograms)>(alignedPointers[2]);
    buf1 = static_cast<decltype(buf1)>(alignedPointers[3]);
    idxBuf1 = static_cast<decltype(idxBuf1)>(alignedPointers[4]);
    buf2 = static_cast<decltype(buf2)>(alignedPointers[5]);
    idxBuf2 = static_cast<decltype(idxBuf2)>(alignedPointers[6]);

    airTopPInitialize<T, IdxT, AccT, BitsPerPass, THREADS_PER_CTA_TOP_P_INIT>
        <<<batchSize, THREADS_PER_CTA_TOP_P_INIT, 0, stream>>>(counters, batchSize, vocabSize, logProbs, nullptr,
            maxTopP, topPs, curandstate, histograms, countHistograms);
    sync_check_cuda_error();

    dim3 grid(blockNum, batchSize);
    // Sample with Top P given sorted tokens
    int constexpr numPasses = calcNumPasses<T, BitsPerPass>();
    auto kernel = airTopPSsampling<T, IdxT, AccT, BitsPerPass, SAMPLING_BLOCK_SIZE, false>;

    for (int pass = 0; pass < numPasses; ++pass)
    {
        if (pass == numPasses - 1)
        {
            kernel = airTopPSsampling<T, IdxT, AccT, BitsPerPass, SAMPLING_BLOCK_SIZE, true>;
        }

        kernel<<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(counters, histograms, countHistograms, outputIds,
            sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, endIds, batchSize, skipDecode,
            pass, buf1, idxBuf1, buf2, idxBuf2);
        sync_check_cuda_error();
    }
}

template void invokeBatchAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    float const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const maxTopP, float const* topPs, hipStream_t stream, int blockNum,
    bool const* skipDecode);

template void invokeBatchAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    half const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const maxTopP, float const* topPs, hipStream_t stream, int blockNum,
    bool const* skipDecode);

template <typename T>
void invokeAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    T const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds,
    float const topP, hipStream_t stream, int blockNum, bool const* skipDecode)
{
    invokeBatchAirTopPSampling(workspace, workspaceSize, outputIds, sequenceLength, finishedInput, finishedOutput,
        cumLogProbs, outputLogProbs, logProbs, curandstate, batchSize, vocabSizePadded, endIds, topP, nullptr, stream,
        blockNum, skipDecode);
}

template void invokeAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    float const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const topP, hipStream_t stream, int blockNum, bool const* skipDecode);

template void invokeAirTopPSampling(void* workspace, size_t& workspaceSize, int** outputIds, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    half const* logProbs, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const topP, hipStream_t stream, int blockNum, bool const* skipDecode);

template unsigned calcAirTopPBlockNum<float, int, float>(int batchSize, int len, int smCnt);
template unsigned calcAirTopPBlockNum<half, int, float>(int batchSize, int len, int smCnt);

} // namespace kernels
} // namespace tensorrt_llm
