#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void topPInitialize(
    int* topPIdValBuf, int* topPOffsetBuf, int* beginTopPOffsetBuf, int const batchSize, int const vocabSize)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    if (bid == 0)
    {
        for (int i = tid; i < batchSize + 1; i += blockDim.x)
        {
            // Inclusive sum of offsets to vocab rows
            topPOffsetBuf[i] = i * vocabSize;
            beginTopPOffsetBuf[i] = topPOffsetBuf[i];
        }
    }

    int index = tid + bid * blockDim.x;

    while (index < batchSize * vocabSize)
    {
        // Set value at {bi, vi} position to vi
        topPIdValBuf[index] = index % vocabSize;
        index += blockDim.x * gridDim.x;
    }
}

void invokeTopPInitialize(int* topPIdValBuf, int* topPOffsetBuf, int* beginTopPOffsetBuf, size_t const batchSize,
    int const vocabSize, hipStream_t stream)
{
    // vocabSize: the column number of logits_buffer for top_p sampling
    topPInitialize<<<32, 512, 0, stream>>>(topPIdValBuf, topPOffsetBuf, beginTopPOffsetBuf, batchSize, vocabSize);
}

template <typename T, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void topPBeamTopKKernel(T const* logProbs, // prob.
    int* topKTmpIdBuf, T* topKTmpValBuf, FinishedState const* finishedInput, int const vocabSize, int* offsetBuf,
    int* beginOffsetBuf, float const topP, float const* topPs, bool const* skipDecode)
{
    /**
     * Kernel performs top 1 search and saves the token with largest probability if it exceeds probability threshold
     */
    int constexpr MAX_K = 1;
    int threadId = threadIdx.x;
    int batchId = blockIdx.x;

    // Skip decoding kernel if configured
    if ((skipDecode != nullptr && skipDecode[batchId])
        || (finishedInput != nullptr && finishedInput[batchId].isSkipDecoding()))
    {
        // Required to skip radix sort
        beginOffsetBuf[batchId] += vocabSize;
        return;
    }

    float pThreshold = (topPs != nullptr) ? topPs[batchId] : topP;

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    TopK<T, MAX_K> partial;

    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i)
    {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (int elemId = threadId; elemId < vocabSize; elemId += THREADBLOCK_SIZE)
    {
        int index = elemId + batchId * vocabSize;
        partial.insert(logProbs[index], elemId);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (threadId == 0)
    {
        beginOffsetBuf[batchId] = offsetBuf[batchId];
        T sumProb = (T) (0.0f);

#pragma unroll
        for (int i = 0; i < MAX_K; i++)
        {
            sumProb += total.u[i];
        }

        if ((float) sumProb >= pThreshold)
        {
            beginOffsetBuf[batchId] += vocabSize;
            int index = batchId * vocabSize;

#pragma unroll
            for (int i = 0; i < MAX_K; ++i)
            {
                topKTmpIdBuf[index + i] = total.p[i];
                topKTmpValBuf[index + i] = total.u[i];
            }
        }
    }
}

struct BlockPrefixCallbackOp
{
    // Running prefix
    float running_total;

    // Constructor
    __device__ BlockPrefixCallbackOp(float running_total)
        : running_total(running_total)
    {
    }

    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide
    // scan.
    __device__ float operator()(float block_aggregate)
    {
        float old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template <typename T>
__device__ void epilogue(int batchId, int currentStep, int offset, int** ids, int* sortedIdVals, T* sortedLogProbs,
    float* cumLogProbs, float* outputLogProbs, int const* endIds, int* sequenceLengths, FinishedState* finishedOutput)
{
    ids[batchId][currentStep] = sortedIdVals[offset];

    if (cumLogProbs != nullptr || outputLogProbs != nullptr)
    {
        float lprob = logf(sortedLogProbs[offset]);
        if (cumLogProbs != nullptr)
        {
            cumLogProbs[batchId] += lprob;
        }
        if (outputLogProbs != nullptr)
        {
            outputLogProbs[batchId] = lprob;
        }
    }
    if (sequenceLengths != nullptr && finishedOutput != nullptr)
    {
        if (ids[batchId][currentStep] == endIds[batchId])
        {
            finishedOutput[batchId].setFinishedEOS();
            // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be outputted
        }
        else
        {
            // We don't need to set output finished state as it is assumed to be in non finished state
            sequenceLengths[batchId] += 1;
        }
    }
}

template <typename T, int blockSize>
__global__ void topPSsampling(T* sortedLogProbs, int* sortedIdVals, int** ids, int* sequenceLength,
    FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs,
    int const* beginOffsetBuf, int const* offsetBuf, int const vocabSize, hiprandState_t* curandstate, float const topP,
    float const* topPs, int const* endIds, int const batchSize, bool const* skipDecode)
{
    /**
     * Each block processes one request row sorted in descending order by probabilities.
     * All threads within block compute running sum of probabilities until one of the threads exceeds the randomly
     * chosen probability threshold. Thread that crossed probaility threshold writes the corresponding token to the
     * output.
     */

    __shared__ float randNumS;

    int const tid = threadIdx.x;
    int const batchId = blockIdx.x;
    // Skip kernel if this sampling method is not chosen
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchId] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchId]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    // Exit early if sequence has finished
    if (finishState.isFinished())
    {
        if (tid == 0)
        {
            if (finishedOutput != nullptr)
            {
                finishedOutput[batchId] = finishState;
            }
            ids[batchId][sequenceLength[batchId]] = endIds[batchId];
        }
        return;
    }

    int constexpr WARP_SIZE = 32;
    int constexpr NUM_WARPS = blockSize / WARP_SIZE;
    int const laneId = threadIdx.x % WARP_SIZE;
    int const warpId = threadIdx.x / WARP_SIZE;
    float const probThreshold = (topPs != nullptr) ? topPs[batchId] : topP;
    int const currentStep = sequenceLength[batchId];

    // With P in (0.0; 1.0] we draw a random number P' in range (0.0; P]
    // We will sum all probs moving from the largest probability to the smallest and
    // will choose the token which probability makes cumulative probability sum to exceed P'
    if (threadIdx.x == 0)
    {
        randNumS = hiprand_uniform(curandstate + blockIdx.x) * probThreshold;
    }

    // if beginOffsetBuf and offsetBuf of sorting have same value,
    // this means that we have find best one in topPBeamTopKKernel
    // So, we can skip this sampling.
    if (beginOffsetBuf[batchId] == offsetBuf[batchId])
    {
        if (tid == 0)
        {
            int offset = batchId * vocabSize;
            epilogue(batchId, currentStep, offset, ids, sortedIdVals, sortedLogProbs, cumLogProbs, outputLogProbs,
                endIds, sequenceLength, finishedOutput);
        }
        return;
    }

    typedef hipcub::BlockScan<float, blockSize> BlockScan;
    __shared__ typename BlockScan::TempStorage tempStorage;
    __shared__ uint32_t selectedShared[NUM_WARPS];
    // Initialize running total
    BlockPrefixCallbackOp prefixOp(0);

    if (laneId == 0)
    {
        selectedShared[warpId] = 0;
    }

    __syncthreads();

    int offset = batchId * vocabSize;
    ids[batchId][currentStep] = sortedIdVals[offset];
    int end = ((vocabSize + blockSize - 1) / blockSize) * blockSize;
    int selectedTokenId = 0;
    // Cumulative sum
    float threadOffset = 0;
    int count = 0;
    for (int vi = tid; vi < end; vi += blockSize)
    {
        float threadProb = (vi < vocabSize) ? (float) sortedLogProbs[offset + vi] : 0.f;
        BlockScan(tempStorage).InclusiveSum(threadProb, threadOffset, prefixOp);
        count = __syncthreads_count(randNumS <= threadOffset);
        selectedTokenId = vi;
        if (count != 0)
        {
            break;
        }
    }

    // select first thread exceeded the prob threshold or the last thread in case of P=1.0f
    if (threadIdx.x == min(blockDim.x - count, blockDim.x - 1))
    {
        epilogue(batchId, currentStep, offset + selectedTokenId, ids, sortedIdVals, sortedLogProbs, cumLogProbs,
            outputLogProbs, endIds, sequenceLength, finishedOutput);
    }
}

template <typename T>
void invokeBatchTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize, int** outputIds,
    int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, T const* logProbs, int const* idVals, int* offsetBuf, int* beginOffsetBuf,
    hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds,
    float const maxTopP, float const* topPs, hipStream_t stream, bool const* skipDecode)
{
    int const vocabSize = vocabSizePadded;

    size_t sortedLogProbBufSize = batchSize * vocabSize * sizeof(T);  // type T
    size_t sortedIdValsBufSize = batchSize * vocabSize * sizeof(int); // type int
    sortedLogProbBufSize = divUp(sortedLogProbBufSize, 256) * 256;
    sortedIdValsBufSize = divUp(sortedIdValsBufSize, 256) * 256;

    void* cubTempStorage = workspace;
    T* sortedLogProbs = (T*) ((char*) cubTempStorage + cubTempStorageSize);
    int* sortedIdVals = (int*) ((char*) sortedLogProbs + sortedLogProbBufSize);

    if (workspace == nullptr)
    {
        check_cuda_error(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, cubTempStorageSize, logProbs,
            (T*) nullptr, idVals, (int*) nullptr, vocabSize * batchSize, batchSize, beginOffsetBuf, offsetBuf + 1,
            0,             // begin_bit
            sizeof(T) * 8, // end_bit = sizeof(KeyT) * 8
            stream));      // hipStream_t
        cubTempStorageSize = divUp(cubTempStorageSize, 256) * 256;
        workspaceSize = sortedLogProbBufSize + sortedIdValsBufSize + cubTempStorageSize;
        return;
    }

    int constexpr BLOCK_SIZE = 256;
    // Performs Top K=1 search.
    // If the most probable token exceeds P, we skip sorting by setting beginOffsetBuf[bi] = offsetBuf[bi]
    topPBeamTopKKernel<T, BLOCK_SIZE><<<batchSize, BLOCK_SIZE, 0, stream>>>(logProbs, sortedIdVals, sortedLogProbs,
        finishedInput, vocabSize, offsetBuf, beginOffsetBuf, maxTopP, topPs, skipDecode);

    // Sort tokens by probability in descending order
    check_cuda_error(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(cubTempStorage, cubTempStorageSize, logProbs,
        sortedLogProbs, idVals, sortedIdVals, vocabSize * batchSize, batchSize, beginOffsetBuf, offsetBuf + 1,
        0,             // begin_bit
        sizeof(T) * 8, // end_bit = sizeof(KeyT) * 8
        stream));      // hipStream_t

    int constexpr SAMPLING_BLOCK_SIZE = 256;
    dim3 grid(batchSize);
    // Sample with Top P given sorted tokens
    topPSsampling<T, SAMPLING_BLOCK_SIZE><<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(sortedLogProbs, sortedIdVals,
        outputIds, sequenceLength, finishedInput, finishedOutput, cumLogProbs, outputLogProbs, beginOffsetBuf,
        offsetBuf + 1, vocabSize, curandstate, maxTopP, topPs, endIds, batchSize, skipDecode);
}

template void invokeBatchTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize,
    int** outputIds, int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput,
    float* cumLogProbs, float* outputLogProbs, float const* logProbs, int const* idVals, int* offsetBuf,
    int* beginOffsetBuf, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const maxTopP, float const* topPs, hipStream_t stream, bool const* skipDecode);

template void invokeBatchTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize,
    int** outputIds, int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput,
    float* cumLogProbs, float* outputLogProbs, half const* logProbs, int const* idVals, int* offsetBuf,
    int* beginOffsetBuf, hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded,
    int const* endIds, float const maxTopP, float const* topPs, hipStream_t stream, bool const* skipDecode);

template <typename T>
void invokeTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize, int** outputIds,
    int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, T const* logProbs, int const* idVals, int* offsetBuf, int* beginOffsetBuf,
    hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds, float const topP,
    hipStream_t stream, bool const* skipDecode)
{
    invokeBatchTopPSampling(workspace, workspaceSize, cubTempStorageSize, outputIds, sequenceLength, finishedInput,
        finishedOutput, cumLogProbs, outputLogProbs, logProbs, idVals, offsetBuf, beginOffsetBuf, curandstate,
        batchSize, vocabSizePadded, endIds, topP, nullptr, stream, skipDecode);
}

template void invokeTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize, int** outputIds,
    int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, float const* logProbs, int const* idVals, int* offsetBuf, int* beginOffsetBuf,
    hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds, float const topP,
    hipStream_t stream, bool const* skipDecode);

template void invokeTopPSampling(void* workspace, size_t& workspaceSize, size_t& cubTempStorageSize, int** outputIds,
    int* sequenceLength, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, half const* logProbs, int const* idVals, int* offsetBuf, int* beginOffsetBuf,
    hiprandState_t* curandstate, int const batchSize, size_t const vocabSizePadded, int const* endIds, float const topP,
    hipStream_t stream, bool const* skipDecode);

__global__ void computeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, int const** outputIds,
    float const* topPDecay, float const* topPMin, int32_t const* topPResetIds, int const* sequenceLengths)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto const currentStep{sequenceLengths[idx]};
    if (outputIds[idx][currentStep] == topPResetIds[idx])
    {
        runtimeTopP[idx] = runtimeInitialTopP[idx];
    }
    else
    {
        runtimeTopP[idx] = max(runtimeTopP[idx] * topPDecay[idx], topPMin[idx]);
    }
}

void invokeComputeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, int const** outputIds,
    float const* topPDecay, float const* topPMin, int32_t const* topPResetIds, int const* sequenceLengths,
    int const local_batchSize, hipStream_t stream)
{
    dim3 block(min(local_batchSize, 512));
    dim3 grid((local_batchSize + block.x - 1) / block.x);
    computeToppDecay<<<grid, block, 0, stream>>>(
        runtimeTopP, runtimeInitialTopP, outputIds, topPDecay, topPMin, topPResetIds, sequenceLengths);
}

} // namespace kernels
} // namespace tensorrt_llm
