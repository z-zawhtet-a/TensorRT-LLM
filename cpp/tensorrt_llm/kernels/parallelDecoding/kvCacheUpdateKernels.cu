#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kvCacheUpdateKernels.h"

#include <array>

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"

namespace tensorrt_llm::kernels::parallel_decoding
{

static constexpr int kUpdateKVCacheKernelShmSize = 16384;

template <typename KVCacheBuffer, int MaxLayerCount, typename MoveEltType>
__global__ void updateKVCacheDraftTokenLocationBatchedKernel(std::array<KVCacheBuffer, MaxLayerCount> kvCacheBuffers,
    const int* seqAcceptedDraftTokenOffsets, const IndexType* packedAcceptedDraftTokensIndices,
    const int32_t* pastKeyValueLengths, int rewindDraftTokenCount, int eltCountPerHead)
{
    int seqIdx = blockIdx.x;
    int headIdx = blockIdx.y;
    int layerIdx = blockIdx.z;
    int warpIdx = threadIdx.x / 32;
    int warpCount = blockDim.x / 32;
    int laneIdx = threadIdx.x & 0x1f;
    int seqDraftTokenStart = seqAcceptedDraftTokenOffsets[seqIdx];
    int seqDraftTokenEnd = seqAcceptedDraftTokenOffsets[seqIdx + 1];
    int seqDraftCount = seqDraftTokenEnd - seqDraftTokenStart;
    if (seqDraftCount == 0)
    {
        return;
    }
    KVCacheBuffer& kvCacheBuffer = kvCacheBuffers[layerIdx];
    int tokenStartIdx = pastKeyValueLengths[seqIdx] - rewindDraftTokenCount;
    int maxEltCountPerMove = kUpdateKVCacheKernelShmSize / sizeof(MoveEltType) / seqDraftCount;
    int eltCountPerMove = min(maxEltCountPerMove, eltCountPerHead);
    __shared__ char loadSmemBuffer[kUpdateKVCacheKernelShmSize];
    auto* eltLoadSmemBuffer = reinterpret_cast<MoveEltType*>(&loadSmemBuffer[0]);
    for (int startChannelOffset = 0; startChannelOffset < eltCountPerHead; startChannelOffset += eltCountPerMove)
    {
        int eltCountCurrentMove = min(eltCountPerMove, eltCountPerHead - startChannelOffset);
        // load K
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = packedAcceptedDraftTokensIndices[seqDraftTokenStart + tokenIdx];
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* kPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getKBlockPtr(seqIdx, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                tokenSmemBuffer[loadChannelIdx] = kPtr[kvLocationIdx];
            }
        }
        __syncthreads();
        // store K
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = tokenIdx;
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* kPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getKBlockPtr(seqIdx, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                kPtr[kvLocationIdx] = tokenSmemBuffer[loadChannelIdx];
            }
        }
        __syncthreads();
        // load V
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = packedAcceptedDraftTokensIndices[seqDraftTokenStart + tokenIdx];
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* vPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getVBlockPtr(seqIdx, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                tokenSmemBuffer[loadChannelIdx] = vPtr[kvLocationIdx];
            }
        }
        __syncthreads();
        // store V
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = tokenIdx;
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenPos * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* vPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getVBlockPtr(seqIdx, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                vPtr[kvLocationIdx] = tokenSmemBuffer[loadChannelIdx];
            }
        }
        __syncthreads();
    }
}

template <typename KVCacheBuffer, int MaxLayerCount>
void updateKVCacheDraftTokenLocationBatched(const KVCacheBuffer* kvCacheBuffers,
    const int* seqAcceptedDraftTokenOffsets, const IndexType* packedAcceptedDraftTokensIndices,
    const int32_t* pastKeyValueLengths, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCount, hipStream_t stream)
{
    // make sure launch buffer is enough
    static_assert(MaxLayerCount * sizeof(KVCacheBuffer) <= 3072);
    if (seqCount == 0 || layerCount == 0)
    {
        return;
    }
    int alignedBytes = 16;
    while (alignedBytes > 0 && (sizeInBytesPerKVHead % alignedBytes != 0))
    {
        alignedBytes >>= 1;
    }
    TLLM_CHECK_WITH_INFO(alignedBytes > 0, "alignedByte should be positive");
    int eltCountPerHead = sizeInBytesPerKVHead / alignedBytes;
    dim3 grid(seqCount, numKVHeads, layerCount);
    dim3 block(128, 1, 1);
    std::array<KVCacheBuffer, MaxLayerCount> kvCacheBufferArray;
    for (int i = 0; i < layerCount; i++)
    {
        kvCacheBufferArray[i] = kvCacheBuffers[i];
    }
    void (*pKernelFunc)(
        std::array<KVCacheBuffer, MaxLayerCount>, const int*, const IndexType*, const int32_t*, int, int)
        = nullptr;
    switch (alignedBytes)
    {
    case 16:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int4>;
        break;
    }
    case 8:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int64_t>;
        break;
    }
    case 4:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int32_t>;
        break;
    }
    case 2:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int16_t>;
        break;
    }
    default:
    {
        TLLM_CHECK_WITH_INFO(alignedBytes == 1, "Strange alignedBytes");
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int8_t>;
        break;
    }
    }
    pKernelFunc<<<grid, block, 0, stream>>>(kvCacheBufferArray, seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, rewindDraftTokenCount, eltCountPerHead);
    TLLM_CUDA_CHECK(hipGetLastError());
}

/*!
 * Update KV cache for parallel decoding algorithms.
 * In following examples, we assume we have 2 sequences, accepted count is [3, 2]
 * @tparam KVCacheBuffer : Type of KV cache, should be LinearKVCache or KVBlockArray
 * @param kvCacheBuffers : list of KVCacheBuffer object
 * @param seqAcceptedDraftTokenOffsets : Array of length seqCount + 1, like [0, 3, 5]
 * @param packedAcceptedDraftTokensIndices : Array of length seqAcceptedDraftTokenOffsets[seqCount], each value is in
 * range [0, maxDraftTokenCount - 1]
 * @param pastKeyValueLengths : Array of length seqCount, meaning how many tokens are already in KV cache
 * @param seqCount : Count of sequence
 * @param numKVHeads : Number of KV heads
 * @param sizeInBytesPerKVHead :
 * @param rewindDraftTokenCount
 * @param stream
 */
template <typename KVCacheBuffer>
void updateKVCacheDraftTokenLocation(const KVCacheBuffer* kvCacheBuffers, const int* seqAcceptedDraftTokenOffsets,
    const IndexType* packedAcceptedDraftTokensIndices, const int32_t* pastKeyValueLengths, int layerCount, int seqCount,
    int numKVHeads, int sizeInBytesPerKVHead, int rewindDraftTokenCount, hipStream_t stream)
{
    int startLayer = 0;
    static constexpr int kMaxLayersPerIter = 32;
    while (startLayer < layerCount)
    {
        int microBatchLayerCount = std::min(layerCount - startLayer, kMaxLayersPerIter);
        updateKVCacheDraftTokenLocationBatched<KVCacheBuffer, kMaxLayersPerIter>(kvCacheBuffers + startLayer,
            seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices, pastKeyValueLengths, microBatchLayerCount,
            seqCount, numKVHeads, sizeInBytesPerKVHead, rewindDraftTokenCount, stream);
        startLayer += microBatchLayerCount;
    }
}

void updateLinearKVCacheDraftTokenLocation(const int* seqAcceptedDraftTokenOffsets,
    const IndexType* packedAcceptedDraftTokensIndices, const int32_t* pastKeyValueLengths,
    int8_t* const* pastKeyValueList, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCount, int maxKVCacheLen, hipStream_t stream)
{
    std::vector<KVLinearBuffer> kvLinearBuffers;
    kvLinearBuffers.reserve(layerCount);
    int sizePerToken = numKVHeads * sizeInBytesPerKVHead;
    for (int i = 0; i < layerCount; i++)
    {
        kvLinearBuffers.emplace_back(seqCount, 0, maxKVCacheLen, sizePerToken, maxKVCacheLen, 0, false);
        kvLinearBuffers.back().data = pastKeyValueList[i];
    }
    updateKVCacheDraftTokenLocation(kvLinearBuffers.data(), seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCount, stream);
}

void updateKVBlockArrayDraftTokenLocation(const int* seqAcceptedDraftTokenOffsets,
    const IndexType* packedAcceptedDraftTokensIndices, const int32_t* pastKeyValueLengths, int64_t* const* pointerArray,
    int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead, int rewindDraftTokenCount,
    int maxKVCacheLen, int maxBlocksPerSeq, int tokensPerBlock, hipStream_t stream)
{
    std::vector<KVBlockArray> kvBlockArrays;
    kvBlockArrays.reserve(layerCount);
    int sizePerToken = numKVHeads * sizeInBytesPerKVHead;
    for (int i = 0; i < layerCount; i++)
    {
        kvBlockArrays.emplace_back(seqCount, maxBlocksPerSeq, tokensPerBlock, sizePerToken, maxKVCacheLen, 0, false);
        kvBlockArrays.back().data = pointerArray[i];
    }
    updateKVCacheDraftTokenLocation(kvBlockArrays.data(), seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCount, stream);
}

} // namespace tensorrt_llm::kernels::parallel_decoding
